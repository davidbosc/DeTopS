#include "hip/hip_runtime.h"
/*
====================================================================================================
 Name        : DeTopS.cu
 Author      : Jesse Harder
 Supervisor  : Dr. Christopher Henry, P. Eng.
 Date        : Sept 16, 2018
 Version     : 2.1
 Modified    : Jesse Harder
 Description : This program will:
                    -Optionally discretize input data, from multiple files
                    -Develop set descriptions for each input set
                    -Perform the descriptive intersection power set for the set of input files (CPU or GPU)
                    -Calculate a measure of the closeness of the sets intersected
                    -Output the results of the intersections to a text file
License     : Licensed under the Non-Profit Open Software License version 3.0
 1) Grant of Copyright License. Licensor grants You a worldwide, royalty-free,
 non-exclusive, sublicensable license, for the duration of the copyright, to do the following:

 a) to reproduce the Original Work in copies, either alone or as part of a collective work;

 b) to translate, adapt, alter, transform, modify, or arrange the Original Work, thereby
 creating derivative works ("Derivative Works") based upon the Original Work;

 c) to distribute or communicate copies of the Original Work and Derivative Works
 to the public, with the proviso that copies of Original Work or Derivative Works
 that You distribute or communicate shall be licensed under this Non-Profit Open Software
 License or as provided in section 17(d);

 d) to perform the Original Work publicly; and

 e) to display the Original Work publicly.

 2) Grant of Patent License. Licensor grants You a worldwide, royalty-free,
 non-exclusive, sublicensable license, under patent claims owned or controlled by
  the Licensor that are embodied in the Original Work as furnished by the Licensor,
  for the duration of the patents, to make, use, sell, offer for sale, have made,
   and import the Original Work and Derivative Works.

 3) Grant of Source Code License. The term "Source Code" means the preferred
 form of the Original Work for making modifications to it and all available
 documentation describing how to modify the Original Work. Licensor agrees to
 provide a machine-readable copy of the Source Code of the Original Work along
 with each copy of the Original Work that Licensor distributes. Licensor reserves
 the right to satisfy this obligation by placing a machine-readable copy of the
 Source Code in an information repository reasonably calculated to permit
 inexpensive and convenient access by You for as long as Licensor continues
 to distribute the Original Work.

 4) Exclusions From License Grant. Neither the names of Licensor, nor the names
 of any contributors to the Original Work, nor any of their trademarks or service
 marks, may be used to endorse or promote products derived from this Original Work
 without express prior permission of the Licensor. Except as expressly stated
  herein, nothing in this License grants any license to Licensor's trademarks,
  copyrights, patents, trade secrets or any other intellectual property. No patent
  license is granted to make, use, sell, offer for sale, have made, or import embodiments
  of any patent claims other than the licensed claims defined in Section 2. No license
 is granted to the trademarks of Licensor even if such marks are included in the Original
  Work. Nothing in this License shall be interpreted to prohibit Licensor from licensing
  under terms different from this License any Original Work that Licensor otherwise would
  have a right to license.

 5) External Deployment. The term "External Deployment" means the use, distribution, or
 communication of the Original Work or Derivative Works in any way such that the Original
 Work or Derivative Works may be used by anyone other than You, whether those works are
 distributed or communicated to those persons or made available as an application intended
 for use over a network. As an express condition for the grants of license hereunder,
 You must treat any External Deployment by You of the Original Work or a Derivative
 Work as a distribution under section 1(c).

 6) Attribution Rights. You must retain, in the Source Code of any Derivative Works
 that You create, all copyright, patent, or trademark notices from the Source Code of
 the Original Work, as well as any notices of licensing and any descriptive text
 identified therein as an "Attribution Notice." You must cause the Source Code for
 any Derivative Works that You create to carry a prominent Attribution Notice reasonably
 calculated to inform recipients that You have modified the Original Work.

 7) Warranty of Provenance and Disclaimer of Warranty. The Original Work is provided
 under this License on an "AS IS" BASIS and WITHOUT WARRANTY, either express or implied,
 including, without limitation, the warranties of non-infringement, merchantability or
 fitness for a particular purpose. THE ENTIRE RISK AS TO THE QUALITY OF THE ORIGINAL WORK
 IS WITH YOU. This DISCLAIMER OF WARRANTY constitutes an essential part of this License.
 No license to the Original Work is granted by this License except under this disclaimer.

 8) Limitation of Liability. Under no circumstances and under no legal theory, whether
 in tort (including negligence), contract, or otherwise, shall the Licensor be liable
 to anyone for any direct, indirect, special, incidental, or consequential damages of
 any character arising as a result of this License or the use of the Original Work
 including, without limitation, damages for loss of goodwill, work stoppage, computer
 failure or malfunction, or any and all other commercial damages or losses. This limitation
 of liability shall not apply to the extent applicable law prohibits such limitation.

 9) Acceptance and Termination. If, at any time, You expressly assented to this License,
 that assent indicates your clear and irrevocable acceptance of this License and all of
 its terms and conditions. If You distribute or communicate copies of the Original Work
 or a Derivative Work, You must make a reasonable effort under the circumstances to obtain
 the express assent of recipients to the terms of this License. This License conditions
 your rights to undertake the activities listed in Section 1, including your right to create
 Derivative Works based upon the Original Work, and doing so without honoring these terms and
 conditions is prohibited by copyright law and international treaty. Nothing in this License
 is intended to affect copyright exceptions and limitations (including "fair use" or "fair
 dealing"). This License shall terminate immediately and You may no longer exercise any of
 the rights granted to You by this License upon your failure to honor the conditions in Section 1(c).

 10) Termination for Patent Action. This License shall terminate automatically and You
 may no longer exercise any of the rights granted to You by this License as of the date
 You commence an action, including a cross-claim or counterclaim, against Licensor or any
 licensee alleging that the Original Work infringes a patent. This termination provision
 shall not apply for an action alleging patent infringement by combinations of the Original
  Work with other software or hardware.

 11) Jurisdiction, Venue and Governing Law. Any action or suit relating to this License
 may be brought only in the courts of a jurisdiction wherein the Licensor resides or in
 which Licensor conducts its primary business, and under the laws of that jurisdiction
 excluding its conflict-of-law provisions. The application of the United Nations Convention
 on Contracts for the International Sale of Goods is expressly excluded. Any use of the Original
 Work outside the scope of this License or after its termination shall be subject to the
 requirements and penalties of copyright or patent law in the appropriate jurisdiction.
 This section shall survive the termination of this License.

 12) Attorneys' Fees. In any action to enforce the terms of this License or seeking
 damages relating thereto, the prevailing party shall be entitled to recover its costs and
 expenses, including, without limitation, reasonable attorneys' fees and costs incurred in
 connection with such action, including any appeal of such action. This section shall survive
 the termination of this License.

 13) Miscellaneous. If any provision of this License is held to be unenforceable, such provision
 shall be reformed only to the extent necessary to make it enforceable.

 14) Definition of "You" in This License. "You" throughout this License, whether in upper or
 lower case, means an individual or a legal entity exercising rights under, and complying with
 all of the terms of, this License. For legal entities, "You" includes any entity that controls,
 is controlled by, or is under common control with you. For purposes of this definition, "control"
 means (i) the power, direct or indirect, to cause the direction or management of such entity,
 whether by contract or otherwise, or (ii) ownership of fifty percent (50%) or more of the outstanding
 shares, or (iii) beneficial ownership of such entity.

 15) Right to Use. You may use the Original Work in all ways not otherwise restricted or conditioned
  by this License or by law, and Licensor promises not to interfere with or be responsible for such uses by You.

 16) Modification of This License. This License is Copyright © 2005 Lawrence Rosen.
 Permission is granted to copy, distribute, or communicate this License without modification.
 Nothing in this License permits You to modify this License as applied to the Original Work or to
 Derivative Works. However, You may modify the text of this License and copy, distribute or communicate
 your modified version (the "Modified License") and apply it to other original works of authorship
 subject to the following conditions: (i) You may not indicate in any way that your Modified License
 is the "Open Software License" or "OSL" and you may not use those names in the name of your Modified
 License; (ii) You must replace the notice specified in the first paragraph above with the notice
 "Licensed under <insert your license name here>" or with a notice of your own that is not confusingly
 similar to the notice in this License; and (iii) You may not claim that your original works are open
 source software unless your Modified License has been approved by Open Source Initiative (OSI) and
 You comply with its license review and certification process.

 17) Non-Profit Amendment. The name of this amended version of the Open Software License ("OSL 3.0")
 is "Non-Profit Open Software License 3.0". The original OSL 3.0 license has been amended as follows:

 (a) Licensor represents and declares that it is a not-for-profit organization that derives no revenue
 whatsoever from the distribution of the Original Work or Derivative Works thereof, or from support
 or services relating thereto.

 (b) The first sentence of Section 7 ["Warranty of Provenance"] of OSL 3.0 has been stricken. For
 Original Works licensed under this Non-Profit OSL 3.0, LICENSOR OFFERS NO WARRANTIES WHATSOEVER.

 (c) In the first sentence of Section 8 ["Limitation of Liability"] of this Non-Profit OSL 3.0,
 the list of damages for which LIABILITY IS LIMITED now includes "direct" damages.

 (d) The proviso in Section 1(c) of this License now refers to this "Non-Profit Open Software
 License" rather than the "Open Software License". You may distribute or communicate the Original
 Work or Derivative Works thereof under this Non-Profit OSL 3.0 license only if You make the
 representation and declaration in paragraph (a) of this Section 17. Otherwise, You shall distribute or
 communicate the Original Work or Derivative Works thereof only under the OSL 3.0 license and You shall
 publish clear licensing notices so stating. Also by way of clarification, this License does not authorize
 You to distribute or communicate works under this Non-Profit OSL 3.0 if You received them under
 the original OSL 3.0 license.

 (e) Original Works licensed under this license shall reference "Non-Profit OSL 3.0"
 in licensing notices to distinguish them from works licensed under the original OSL 3.0 license.
====================================================================================================
*/
//---------------------------------------------------------------------------------
#include <iostream>   //Standard input output
#include <fstream>    //Read input and write output files
#include <vector>     //Provides access to vector object, for flexibly sized arrays
#include <math.h>     //Provides math functions. pow, log, ceil, floor
#include <stdlib.h>   //Provides size_t datatype
#include <string>     //Provides string object
#include <sstream>    //Provides methods for working with strings
#include <limits>     //Used to derive minFloat
#include <ctime>      //Used for CPU timing code
#include <pthread.h>  //Used for parallel CPU threads
#include <mutex>      //Used for synchronization of parallel cpu code
//---------------------------------------------------------------------------------
static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

unsigned SETS = 10;    //How many subsets to load in (for testing)
#define STREAMS 500    //How many streams to launch intersectKernels in
typedef unsigned long long bitString;

bool emptySetCheck = false;
//Most negative float value, used as a null in arrays
const float minFloat = (-1) * (std::numeric_limits<float>::max());
//Maximum depth of intersections (max number of sets that can take place in an intersection)
unsigned maxDepth = 0;       
unsigned F_SUBSET_COUNT = 0;  //Number of input sets
unsigned VECTORS_PER_SUBSET;  //Width of each fundamental subset
unsigned VECTOR_SIZE;         //Features per feature vector, defines shared memory tile length
unsigned WIDTH;               //Total width of the output set
unsigned CORES = 1;           //How many cores to run cpu on
unsigned TILE_WIDTH;          //Tile width of intersectKernel
unsigned SUBSETS_PER_FAMILY;  //Number of subsets within each family
bool PSEUDOMETRIC_USES_DESCRIPTIVE_INTERSECTIONS = true;

//Global variables used for parallel CPU intersection code
bitString bitPermute;
bitString bitCount;
unsigned cpuDepth = 0;
std::mutex mtx;

using namespace std;

/**
 * This structure is used for passing multiple arguments to the 
 * CPU Intersection function (intersectCPU)
 *     prefixes: A prefix summed set of a row of Pascal's Triangle
 *       pascal: The current pascal number (how many intersections to perform)
 *            a: A pointer to the intersections array (input and output)
 */
typedef struct{
    unsigned pascal;
    float *a;
    float *prefixes;
} intersectArgs;

template<typename T>
using metric_t = T(*) (T*, T*, unsigned, unsigned, unsigned, float);

template<typename T>
using pseudometric_t = T(*) (T*, T*, T*, unsigned, unsigned, unsigned, float, unsigned, unsigned, unsigned, metric_t<T>);

template<typename T>
__host__ __device__ T vectorHammingDistance(
	T* d_A,
	T* d_B,
	unsigned index_A,
	unsigned index_B,
	unsigned VECTOR_SIZE,
	float minFloat
) {
	unsigned distance = 0;
	for (unsigned k = 0; k < VECTOR_SIZE; k++) {
		if (d_A[index_A + k] != minFloat &&
			d_B[index_B + k] != minFloat) {
			if (d_A[index_A + k] != d_B[index_B + k]) {
				distance++;
			}
		}
	}
	return distance;
}

template<typename T>
__host__ __device__ T descJaccardDistance(
	T* A_desc,
	T* B_desc,
	T* desc_intersection,
	unsigned index_A,
	unsigned index_B,
	unsigned size,
	float minFloat,
	unsigned VECTOR_SIZE,
	unsigned VECTORS_PER_SUBSET,
	unsigned SUBSETS_PER_FAMILY,
	metric_t<T> embeddedMetric
) {
	unsigned descriptiveIntersectionCardinality = 0;
	unsigned unionCardinality = 0;

	//starting at index_B * size_A + index_A of the array containing all descriptive intersections
	//(in row major layout), get all the vectors that aren't minFloat
	unsigned desc_intersections_index = index_A * SUBSETS_PER_FAMILY + index_B;

	unsigned subsetAIndex = index_A * VECTOR_SIZE * VECTORS_PER_SUBSET;
	unsigned subsetBIndex = index_B * VECTOR_SIZE * VECTORS_PER_SUBSET;

	unsigned inputSetVectorOffset = desc_intersections_index * VECTOR_SIZE * VECTORS_PER_SUBSET;

	unsigned maxUnionSize;
	unsigned numberOfVectorsInA = 0;
	unsigned numberOfVectorsInB = 0;

	for (int i = 0; i < size; i += VECTOR_SIZE) {
		if (desc_intersection[inputSetVectorOffset + i] != minFloat) {
			descriptiveIntersectionCardinality++;
		}
	}

	//get the number of vectors in the description of A...
	for (int i = 0; i < size; i += VECTOR_SIZE) {
		if (A_desc[subsetAIndex + i] != minFloat) {
			numberOfVectorsInA++;
		}
	}

	//get the number of vectors in the description of B...
	for (int i = 0; i < size; i += VECTOR_SIZE) {
		if (B_desc[subsetBIndex + i] != minFloat) {
			numberOfVectorsInB++;
		}
	}

	maxUnionSize = numberOfVectorsInA + numberOfVectorsInB;

	unionCardinality = maxUnionSize - descriptiveIntersectionCardinality;
	return 1.0f - ((float)descriptiveIntersectionCardinality / (float)unionCardinality);
}

template<typename T>
__host__ __device__ T descHausdorffDistance(
	T* A_desc,
	T* B_desc,
	T* desc_intersection,	//unused
	unsigned index_A,
	unsigned index_B,
	unsigned size,			//unused
	float minFloat,
	unsigned VECTOR_SIZE,
	unsigned VECTORS_PER_SUBSET,
	unsigned SUBSETS_PER_FAMILY,
	metric_t<T> embeddedMetric
) {
	unsigned* distanceBetweenEachVector = new unsigned[VECTORS_PER_SUBSET * VECTORS_PER_SUBSET];
	unsigned* minOfCols = new unsigned[VECTORS_PER_SUBSET];
	unsigned* minOfRows = new unsigned[VECTORS_PER_SUBSET];

	unsigned subsetAIndex = index_A * VECTOR_SIZE * VECTORS_PER_SUBSET;
	unsigned subsetBIndex = index_B * VECTOR_SIZE * VECTORS_PER_SUBSET;

	//Build a matrix of distances
	//for each a in A_i
	for (unsigned i = 0; i < VECTORS_PER_SUBSET; i++) {
		//take the distance with each b in B_j
		for (unsigned j = 0; j < VECTORS_PER_SUBSET; j++) {
			unsigned distance = embeddedMetric(
				A_desc,
				B_desc,
				subsetAIndex + j * VECTOR_SIZE,
				subsetBIndex + i * VECTOR_SIZE,
				VECTOR_SIZE,
				minFloat
			);
			distanceBetweenEachVector[i * VECTORS_PER_SUBSET + j] = distance;
		}
	}

	//Find the min of each row and column
	//for each col
	for (unsigned i = 0; i < VECTORS_PER_SUBSET; i++) {
		//go through each row and find the min
		unsigned minOfCol = distanceBetweenEachVector[i];
		unsigned minOfRow = distanceBetweenEachVector[i * VECTORS_PER_SUBSET];
		for (unsigned j = 1; j < VECTORS_PER_SUBSET; j++) {
			minOfCol = minOfCol < distanceBetweenEachVector[j * VECTORS_PER_SUBSET + i] ?
				minOfCol : distanceBetweenEachVector[j * VECTORS_PER_SUBSET + i];
			minOfCols[i] = minOfCol;

			minOfRow = minOfRow < distanceBetweenEachVector[i * VECTORS_PER_SUBSET + j] ?
				minOfRow : distanceBetweenEachVector[i * VECTORS_PER_SUBSET + j];
			minOfRows[i] = minOfRow;
		}
	}

	//Find the max
	unsigned maxOfMinCols = minOfCols[0];
	unsigned maxOfMinRows = minOfRows[0];
	for (int i = 1; i < VECTORS_PER_SUBSET; i++) {
		maxOfMinCols = maxOfMinCols > minOfCols[i] ?
			maxOfMinCols : minOfCols[i];
		maxOfMinRows = maxOfMinRows > minOfRows[i] ?
			maxOfMinRows : minOfRows[i];
	}

	return max(maxOfMinCols, maxOfMinRows);
}

template <typename T>
__device__ pseudometric_t<T> p_descJaccardDistance = descJaccardDistance<T>;

template <typename T>
__device__ pseudometric_t<T> p_descHausdorffDistance = descHausdorffDistance<T>;

template <typename T>
__device__ metric_t<T> p_no_embeddedMetric;

template <typename T>
__device__ metric_t<T> p_vectorHammingDistance = vectorHammingDistance<T>;


/******************************************************************************
 * isEmptyKernel
 *
 * This function will determine if a set in the intersections set is the empty set
 * If a thread encounters a non (minFloat) value, it writes a 1 to that intersections
 * position in b, indicating that some non-empty results are in the intersection
 * [in]:
 *         a: The set containing the results of all performed intersections
 *         b: A set containing one value for each set/intersection in a
 *         index: The location in a of the subset to be checked
 *         VECTORS_PER_SUBSET: How many values need to be checked in b
 *         minFloat: The most negative float value, signifies a null or empty result
 *
 * [out]:
 *        b: Modified to have a 1 in the position of every non-empty set
 * [return]:
 *        isEmpty: True if the first value of every vector in the subset is minFloat
 *
 *****************************************************************************/
__global__ void isEmptyKernel(float* a, float *b, bitString index, unsigned VECTORS_PER_SUBSET, 
                                  float  minFloat) {

    //Tracks if any thread in block has found a non empty vector
    __shared__ bool isNotEmpty;

    unsigned id = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (a[VECTORS_PER_SUBSET * index + id] > minFloat && id < VECTORS_PER_SUBSET) {
        isNotEmpty = true;
    }
    __syncthreads();

    //if non-empty vector is found, thread 0 writes to the output array
    if(isNotEmpty == true && threadIdx.x == 0){
        b[index] = 1;
    }
}

/******************************************************************************
 * isEmptySet
 *
 * This function will determine if a set in the intersections set is the empty set
 *
 * [in]:
 *         a: The set containing the results of all performed intersections
 *         index: The location in a of the subset to be checked
 *
 * [return]:
 *         isEmpty: True if the first value of every vector in the subset is minFloat
 *
 *****************************************************************************/
 bool isEmptySet(float *a, bitString index){

    for(unsigned i=0; i < VECTORS_PER_SUBSET; i++){
        if(a[index + i] > minFloat){
            return false;
        }
    }
    return true;
}

/******************************************************************************
 *
 * intersectKernel
 *
 * Each thread intersects an object from one set with all of the objects in another set
 * Objects found in both sets are printed out
 *
 * [in]:
 *         *a: An array containing all sets and vectors
 *         F_SUBSET_SIZE: Number of fundamental subsets in the input
 *         VECTORS_PER_SUBSET: The number of objects per set
 *         VECTOR_SIZE: Number of elements in each vector
 *         indexA: The array index for set A
 *         indexB: The array index for set B
 *         minFloat: Lowest float value, used for "null" data
 *
 * [out]:
 *         A set of all feature vectors that appear in both a[index] 
 *         and a given vector in intersectionSet
 *
 * [return]:
 *         Void
 *
 *******************************************************************************/
__global__ void intersectKernel(float *a, unsigned F_SUBSET_COUNT, unsigned VECTORS_PER_SUBSET, 
                                    unsigned VECTOR_SIZE, bitString indexA, bitString indexB, 
                                    bitString indexC, float minFloat, unsigned WIDTH){

    //Shared memory to store the two shared memory matrices, A B
    extern __shared__ float tiles[];
    unsigned TILE_WIDTH = blockDim.x;
    float *tileA = &tiles[0];
    float *tileB = &tiles[(TILE_WIDTH * (VECTOR_SIZE + 1))];

    //Boolean  that tracks if this thread's vector has matched with any vector in the other set
    bool inIntersect = false;
    unsigned tx = threadIdx.x;
    unsigned width = WIDTH;

    //Overall id of current thread
    unsigned id = (blockDim.x * blockIdx.x) + threadIdx.x;

    //Load this thread's vector of a into shared memory
    for(unsigned i = 0; i < VECTOR_SIZE + 1; ++i){
        if(id < VECTORS_PER_SUBSET){
            tileA[i*TILE_WIDTH + tx] = a[indexA * VECTORS_PER_SUBSET + id + width * i];
        }else{
            //Set element to minimum value (value to be ignored), if the thread is out of bounds
            tileA[i*TILE_WIDTH + tx] = minFloat;
        }
    }

    for(unsigned q = 0; q < gridDim.x; ++q){
        //Load this thread's corresponding vector in the qth tile of b into shared memory
        for(unsigned i = 0; i < VECTOR_SIZE + 1; ++i){
            if((q * TILE_WIDTH) + tx < VECTORS_PER_SUBSET){
                tileB[i*TILE_WIDTH + tx] = 
                    a[indexB * VECTORS_PER_SUBSET + tx + (TILE_WIDTH * q) + (width * i)];
            }else{
                //Set element to minimum value (value to be ignored), if the thread is out of bounds
                tileB[i*TILE_WIDTH + tx] = minFloat;
            }
        }

        __syncthreads();

        if(tileA[tx] > minFloat && inIntersect == false){

            for(unsigned i=0; i < TILE_WIDTH; ++i){
                //Two vectors are equal until non-equal elements in the vectors are encountered
                bool match = true;

                for(unsigned j=0; j < VECTOR_SIZE; ++j){
                ////! Replace this if statement with a function to suit your implementation!////
                    if(tileA[j*TILE_WIDTH + tx] != tileB[j*TILE_WIDTH + i]){
                        match = false;
                        break;
                    }
                }

                if(match == true){
                    inIntersect = true;
                    tileA[(VECTOR_SIZE)*TILE_WIDTH + tx] += tileB[(VECTOR_SIZE)*TILE_WIDTH + i];
                    break;
                }
            }

        }
        __syncthreads();
    }

    //If this vector has not matched with any vector in the other set, set it to "null"
    if(inIntersect == false){
        for(unsigned i = 0; i < VECTOR_SIZE + 1; ++i){
            tileA[i*TILE_WIDTH + tx] = minFloat;
        }
    }

    __syncthreads();

    //Write tileA to it's space in a (the intersection power set)
    if(id < VECTORS_PER_SUBSET){
        for(unsigned i = 0; i < VECTOR_SIZE + 1; ++i){
            a[indexC * VECTORS_PER_SUBSET + id + width * i] = tileA[i*TILE_WIDTH + tx];
        }
    }
}

/******************************************************************************
 * getTrailingZeros
 *
 * Calculates how many trailing 0s exists in the binary form of a number
 *         This function is to be called as a part of function: next_perm
 * [in]:
 *         w: Some integer to be checked
 * [out]:
 *         None
 * [return]:
 *         x: The count of the trailing zeros of w
 *
 *******************************************************************************/
bitString getTrailingZeros(bitString w){
    bitString x = 0;
    while(w % 2 == 0 && w > 0){
        w = w >> 1;
        x++;
    }
    return x;
}

/******************************************************************************
 * next_perm
 * Gives the next permutation for a bit sequence containing the same number of bits as v
 * [in]: v: previous permutation
 * [out]: none
 * [return]: Next permutation for a bit sequence containing the same number of bits as v
 * [comments]: Used in conjunction with element_0. Function obtained from:
 *               http://alexbowe.com/popcount-permutations/
 *               http://graphics.stanford.edu/~seander/bithacks.html
 *
 *******************************************************************************/
bitString next_perm(bitString v){
    //v is the current permutation of bits
    bitString w; //next permutation of bits

    bitString t = v | (v - 1); // t gets v's least significant 0 bits set to 1
    //Next set to 1 the most significant bit to change,
    //set to 0 the least significant ones, and add the necessary 1 bits.
    w = (t + 1) | (((~t & -~t) - 1) >> (getTrailingZeros(v) + 1));

    return w;
}

/******************************************************************************
 * getBitPatternIndex
 *
 * This function returns a number corresponding to which bit pattern the input is at a given level
 * For example, 0011 is the first pattern of 2 bits, 0101 the second , 0110 the third...
 *
 *
 * [in]:
 *         depth: the number of on bits in the bit pattern
 *         bitPattern: a string of bits with depth 1's
 *
 * [out]:
 *        None
 *
 * [return]:
 *         i: the index of which combination of #depth bits was provided
 *
 *******************************************************************************/
bitString getBitPatternIndex(unsigned depth, bitString bitPattern){
    unsigned i = 0; //Stores the current index of the bit pattern 
    bitString compareBits = (1 << depth) -1; //Get first bit pattern with #depth one's
    while( true ){
        if(bitPattern == compareBits)
            return i;
        i++;
        compareBits = next_perm(compareBits);
    }
}

/******************************************************************************
 * intersectCPU
 *
 * This function will intersect two sets together using the CPU.
 * Each thread of this function will handle an intersection.
 * When the intersection is completed, the thread will check if there are more intersections to do
 * if not, exit function
 *
 * [in]:
 *         args: An object containing:
 *            a: The intersections set, where data is read from, and written to
 *       pascal: The Pascal number stating how many intersections need to be performed at this level
 *     prefixes: A prefix sum of the pascal numbers, used to find indexes
 * [out]:
 *        This function will insert the result of the intersection to the intersection array
 *        at set index singleBit+myBits
 *
 * [return]:
 *         Void
 *******************************************************************************/
void *intersectCPU(void* args){
    bitString curBits;     //Bit index of the result of the intersection
    bitString myIndex;     //Index of Set A in intersections array
    bitString singleIndex; //Index of Set B in intersections array
    bitString outIndex;    //Index of output Set C in intersections array

    intersectArgs *arg = (intersectArgs*)args; //Holds data structure defined above
    std::unique_lock<std::mutex> critZone (mtx, std::defer_lock); //Declare lock for parallelization

    //Loop runs until there are no more intersections to be performed at this level
    while(true){
    ///////////////only one thread may perform this section at a time////////////////////
        critZone.lock();

        //If all intersections have been performed, exit function
        if(bitCount >= arg->pascal){
            free(arg);
            critZone.unlock();
            return 0;
        }

        //Get the next intersection to be performed
        if(bitCount > 0){
            bitPermute = next_perm(bitPermute);
        }
        curBits = bitPermute;
        outIndex = arg->prefixes[ 1 + cpuDepth ] + bitCount;
        singleIndex = 1+getBitPatternIndex(1, curBits & -curBits);
        myIndex = arg->prefixes[cpuDepth] + 
                      getBitPatternIndex(cpuDepth, curBits - (curBits & -curBits));
        bitCount++;
        critZone.unlock();
    ///////////////end single thread critical section //////////////////////////////////////

        //Determine which sets are being intersected
        outIndex *= VECTORS_PER_SUBSET;
        myIndex *= VECTORS_PER_SUBSET;
        singleIndex *= VECTORS_PER_SUBSET;

        //Tracks how many equal vectors have been found, 
        //This is used as an index for writing matched vectors to the Power Set array
        unsigned count=0;

        //If the set to be intersected is the empty set, skip to next intersection
        if(emptySetCheck == true){
            if(isEmptySet(arg->a, myIndex) == true){
                continue;
            }
        }

        //for each vector in set a
        for(unsigned k = 0; k < VECTORS_PER_SUBSET; ++k){
            //Compare to each vector in set b
            for(unsigned i = 0; i < VECTORS_PER_SUBSET; ++i){
                //Match is true until a value that does not match between the two vectors is found
                bool match = true;
                //Compare all of the elements in the two vectors
                for(unsigned j = 0; j< VECTOR_SIZE; ++j){
                    //If two elements don't match, or the value is minFloat the match is false
                    if(arg->a[singleIndex + k + (WIDTH * j)] != arg->a[myIndex + i + (WIDTH *j)] 
                           || arg->a[singleIndex + k + (WIDTH * j)] <= minFloat){
                        match = false;
                        break;
                    }
                }
                //If no non-matching elements are found, 
                // write the matching vector to the Intersection Power Set array
                if(match == true){
                    for(unsigned m = 0; m < VECTOR_SIZE + 1; ++m){
                        arg->a[outIndex  + k + (WIDTH * m)] =
                            arg->a[singleIndex + k + (WIDTH*m)];

                        //Add the feature counts of the two matching vectors together
                        if(m == VECTOR_SIZE)
                            arg->a[outIndex + k + (WIDTH * m)] =
                                arg->a[singleIndex + k + (WIDTH * m)] +
                                arg->a[myIndex + i + (WIDTH * m)];
                    }
                    count++;
                    break;
                }
            }
        }
    }
}

/******************************************************************************
 * discretize
 *
 * Discretizes an array of floats (Values 1 to (specified number of values) )
 *
 * [in]:
 *         *data: pointer to the array of floats to be discretized
 *         size: the size of the data array
 *         num_bins: the number of discrete values to divide the data into,
 *                   determined by user input
 *
 * [out]:
 *         *data: Discretized array
 *
 * [return]:
 *         void
 *
 *******************************************************************************/
void discretize(float *data, unsigned size, unsigned num_bins) {

    float min = data[0];    //Records the maximum value of the input data
    float max = data[0];    //Records the minimum value of the input data
    float *bounds = new float[num_bins + 1];    //Calculates the boundary values of the array

    //Get max and min of data set
    for (unsigned i = 0; i < size; ++i) {
        if (data[i] < min)
            min = data[i];
        if (data[i] > max) {
            max = data[i];
        }
    }
    
    //If min < 0, then shift values to the right, so all are positive
    if (min < 0) {
        for (unsigned i = 0; i < size; ++i) {
            data[i] += min * (-1);
        }
    }

    //If max != 1, divide all values in range 0-1
    if (max != 1) {
        for (unsigned i = 0; i < size; ++i) {
            data[i] /= max;
        }
    }

    //Generate bounds for partition, based on int parts
    float partSize = 1.0 / num_bins;
    for (unsigned i = 0; i < num_bins + 1; ++i) {
        bounds[i] = i * partSize;
    }

    //Set each data value into it's value range
    for (unsigned i = 0; i < size; ++i) {
        if (data[i] == bounds[num_bins]) {
            data[i] = num_bins;
            continue;
        }
        for (unsigned j = 0; j < num_bins; ++j) {
            if (data[i] >= bounds[j] && data[i] < bounds[j + 1]) {
                data[i] = j + 1;
            }
        }
    }
}

/******************************************************************************
 * initNegative
 *
 * Initializes the values in a float array to the lowest float value
 * These values serve as a check for `null` values in the set descriptions
 * The values in the last row are set to 1`s, the initial value of the object's 
 * frequency in the fundamental subset
 *
 * [in]:
 *         *data: The set description array, to be initialized
 *         size: The number of elements in the array, not including vector counts
 * [out]:
 *         *data: All values initialized to the most nagative float value
 *
 * [return]:
 *         Void
 *
*******************************************************************************/
void initNegative(float *data, unsigned size) {
    
    for (unsigned i = 0; i < size; ++i) {
        data[i] = minFloat;
    }
}

/******************************************************************************
 * createSetDescription
 *
 * Takes in a set of feature vectors, then finds all unique vectors in the set
 *
 *     [in]:
 *         *v: pointer to a set of feature vectors
 *         *w: pointer to output for set of unique descriptions
 *
 *     [out]:
 *         *w: vector of feature vectors to be filled with unique descriptions
 *
 *     [return]:
 *         void
 *
 *******************************************************************************/
void createSetDescription(float *v, float *w){

    for(int fa = 0; fa < F_SUBSET_COUNT; ++fa){
        unsigned setIndex = (fa + 1) * VECTORS_PER_SUBSET;
        int uniqueCount = 0;//Running total of the number of unique objects encountered
        
        //For each vector in A
        for(unsigned i = 0; i < VECTORS_PER_SUBSET; ++i){
            //Tracks if the current vector of v is unique (has not matched with any vectors of w)
            bool isUnique = true;
            //For each vector in D(A)
            for(unsigned j = 0; j < uniqueCount; ++j){
                //Tracks if the current vector of v matches with the current vector of w
                bool unique = false;
                //For each element in current vector
                for(unsigned k = 0; k < VECTOR_SIZE; k++){
                    //If any two elements don't match, then the two vectors don't match
                    if(v[(fa * VECTORS_PER_SUBSET) + (k * VECTORS_PER_SUBSET * F_SUBSET_COUNT) +i]
                          != w[setIndex + (k * WIDTH ) + j]){
                        unique = true;
                        break;
                    }
                }

                if(unique == false){
                    isUnique = false;
                    //If vector is not unique, increment the conut of the vector it matched with
                    w[setIndex + (VECTOR_SIZE * WIDTH ) + j]++;
                    break;
                }
            }
    
            if(isUnique){
                //If the vector is unique, insert it into intersection set
                for(int j = 0; j < VECTOR_SIZE; ++j){
                    w[setIndex + (j * WIDTH) + uniqueCount] = 
                       v[(fa * VECTORS_PER_SUBSET) + (j * VECTORS_PER_SUBSET * F_SUBSET_COUNT) +i];
                }
                w[setIndex + (VECTOR_SIZE * WIDTH) + uniqueCount] = 1;
                uniqueCount++;
            }
        }
    }
}

/******************************************************************************
 * calculateMeasure
 *
 * Calculates the final measure of closeness of sets
 *
 *     [in]:
 *         emptySetSize: the number of sets to be calculated
 *         *prefixPascal: an array of prefix summed Pascal numbers
 *         *intersections: the array of data to be operated upon
 *         pascalTotal: a weighted sum of pascal numbers, used for calculation
 *         verbose_info: a boolean specifying whether to print detailed info or not
 *         measure_within_set: a boolean specifying whether intersections of sets that are all 
 *                             within one family should be included or not
 *
 *     [out]:
 *         none
 *
 *     [return]:
 *         totalMeasure: the final result of the measure calculation
 *
 *******************************************************************************/
float calculateMeasure(unsigned emptySetSize, float* prefixPascal, float* intersections, 
                          float pascalTotal, bool verbose_info, bool measure_within_set){
    float totalMeasure = 0; //Stores total measure
    float weightedOut = 0;  //Total measure lost to intersections we don't want to include
    //how many sets are in each intersection
    unsigned depth = 0;
    unsigned checkPoint = 0;
	
    if(measure_within_set == true){
        printf("Include single family intersections in measure\n");
    }else{
        checkPoint = pow(2, F_SUBSET_COUNT/2);
        printf("Exclude single family intersections from measure: CheckPoint = %i\n", checkPoint);
    }

    for(bitString i = 1; i < emptySetSize; ++i){
        //Total count of vectors in this intersection
        float total = 0;
        bitString bitPattern = 0;
        if(i == prefixPascal[depth+1]){
            depth++;
            bitPattern = (1 << depth) -1;
        }else{
            bitPattern = next_perm(bitPattern);
        }

        //Total all of the counts of vectors in this set
        for(unsigned j = 0; j < VECTORS_PER_SUBSET; ++j){
            if(intersections[(VECTORS_PER_SUBSET * i) + j + (WIDTH * VECTOR_SIZE)] > minFloat){
                total += intersections[(VECTORS_PER_SUBSET * i) + j + WIDTH * VECTOR_SIZE];
            }
        }
        //Calculate the weighted value of this set's count, and add it to the final measure
        float weightedValue = ((float)depth / pascalTotal) * (total / VECTORS_PER_SUBSET);
      
        //Ayotu
        if(measure_within_set == false && (bitPattern<checkPoint || (bitPattern%checkPoint == 0))){
            weightedOut += weightedValue;
            if(verbose_info == true) 
                printf("(Excluded from measure)");
        }else{
            totalMeasure += weightedValue;
        }
        //Print detailed information on each intersection performed
        if(verbose_info == true){
            std::cout << "Bit Pattern :" << bitPattern << ", ";
            printf("Index: %i, #Sets: %i  Count: %f / %i, Weighted: %f\n",
                       i, depth, total, depth * VECTORS_PER_SUBSET, weightedValue);
        }
    }

    //Remove the weight of single family intersections
    totalMeasure /= (1 - weightedOut);
	
    //Account for rounding
    if(totalMeasure > 1) totalMeasure = 1;

    //totalMeasure = (totalMeasure - ((float)F_SUBSET_COUNT / pascalTotal)) / 
    //                   (1 - ((float)F_SUBSET_COUNT / pascalTotal) );
    return totalMeasure;
}

/******************************************************************************
 * writeToFile
 *
 * Writes the results of the intersections to a text file
 *
 * [in]:
 *         *originalValue: an array of the original values read in from the input files
 *         *intersections: the array of data to be operated upon
 *
 *     [out]:
 *         result.txt: A text file containing all of the results from the intersections
 *
 *     [return]:
 *         void
 *
 *     [notes]:
 *         Results written map to the least significant bit (set) that 
 *         was involved in the intersection
 *         ie: Set 1 2 and 3 intersect, output will be vectors from Set 1
 *         Counts of each vector are printed in parentheses at the end of each vector
 *
 *******************************************************************************/
void writeToFile(float *intersections, float *originalValues){

    ofstream out("result.txt"); //Write output of final intersection to file
    unsigned curPascal = F_SUBSET_COUNT;//Tracks many sets exist in each level of depth(Inital: 1 set at depth 0)   
    unsigned intersectIndex = 1; //An overall count of which intersection is being written

    for(unsigned k = 1; k <= maxDepth; k++){
        for(bitString j = 0; j < curPascal; ++j){
            //A bit pattern showing which sets were invloved in the intersectIndex'th intersection
            bitString bitPattern = 0;

            if(j == 0){
                //Get first pattern of k bits
                bitPattern = (1 << k) -1;
            }else{
                //Get next pattern of k bits
                bitPattern = next_perm(bitPattern);
            }
			//Get the least significant bit from the bitPattern
			//TODO: Make this OS independant 
			//bitString setIndex = __builtin_ffs(bitPattern) - 1;
			unsigned long setIndex;
			unsigned char isNonzero = _BitScanReverse64(&setIndex, bitPattern);
            //Write which set this is, and what bit pattern it maps to
            out << "Set: " << intersectIndex << " Bit pattern: " << bitPattern << 
			    " Least bit: " << setIndex << endl;
            
            for(unsigned i = 0; i < VECTORS_PER_SUBSET * (VECTOR_SIZE + 1); ++i){
                if(intersections[(intersectIndex * VECTORS_PER_SUBSET) + WIDTH * 
                    (i % (VECTOR_SIZE + 1)) + i / (VECTOR_SIZE + 1) ] != minFloat){
                    //If this is the last element, print the vector count from intersections
                    if(i % (VECTOR_SIZE + 1) == (VECTOR_SIZE)){
                        out << "(" << intersections[(intersectIndex * VECTORS_PER_SUBSET) + 
                            WIDTH * (VECTOR_SIZE) + i / (VECTOR_SIZE + 1) ] << ")" << endl;
                    }else{
                        //Write the values, mapped to the original input values
                        out << originalValues[(setIndex * VECTORS_PER_SUBSET) +  
                            (VECTORS_PER_SUBSET * F_SUBSET_COUNT) * (i % (VECTOR_SIZE + 1)) 
                            + i / (VECTOR_SIZE + 1) ] << " ";
                    }
                }
            }
            intersectIndex++;
        }
        //Get the number of sets at the next level of depth
        curPascal = curPascal * ((F_SUBSET_COUNT - k)/ (k + 1.0));
    }
}

/******************************************************************************
 * writeToFile_D
 *
 * Writes the discretized results of intersections to a text file
 *
 * [in]:
 *         *originalValue: an array of the original values read in from the input files
 *         size: The number of sets to be written
 *
 * [out]:
 *         result.txt: A text file containing all of the results from the intersections(discretized)
 *
 * [return]:
 *         void
 *
 * [notes]: The count of how many times a vector appeared in an intersection is written in
 *          parentheses at the end of the vector
 *
 *******************************************************************************/
void writeToFile_D(float *intersections, unsigned size){

    ofstream out("result.txt"); //Write output of final intersection to file

    for(unsigned k = 0; k < size; k++){
        out << "Set " << k << endl;
        for(unsigned i = 0; i < VECTORS_PER_SUBSET * (VECTOR_SIZE + 1); ++i){
            if(intersections[(k * VECTORS_PER_SUBSET) + WIDTH * (i % (VECTOR_SIZE + 1)) + 
                   i / (VECTOR_SIZE + 1)] != minFloat){
                if(i % (VECTOR_SIZE + 1)== (VECTOR_SIZE)){
                    out << "(" << intersections[(k * VECTORS_PER_SUBSET) + WIDTH * 
                        (VECTOR_SIZE) + i / (VECTOR_SIZE + 1) ] << ")" << endl;
                }else{
                    out << intersections[(k * VECTORS_PER_SUBSET) +  WIDTH * 
                        (i % (VECTOR_SIZE + 1)) + i / (VECTOR_SIZE + 1) ] << " ";
                }
            }
        }
    }
}

/******************************************************************************
 * printHelp
 *
 * Prints out all available command parameters, and a short description of each
 *
 * [in]:
 *         None
 * [out]:
 *         A list and description of all command parameters
 *
 * [return]:
 *         void
 *
 *******************************************************************************/
void printHelp(){
    printf("\nCommand Parameters:\n");
    printf("\t-b [int > 0]: Specifies how many bins to discretize into, if discretizing\n");
    printf("\t-c: Instructs program to run all intersections on the CPU\n");
    printf("\t-cg: Instructs program to run all intersections on GPU, then again on CPU (used for testing)\n");
    printf("\t-cores [int >0]: Specifies how many cores to run parallel CPU code on\n");
    printf("\t-d: Instructs program to discretize the input data (Default: 3 bins)\n");
    printf("\t-f [file0 file1 ... fileN]: Manually list all input files to use !!Must be last parameter!!\n");
    printf("\t-fd [int > 0] [file0] [file1]: Specifies to read in X files from exactly 2 file locations, file0 and file1\n");
    printf("\t-gpu [int >= 0]: Specify which device to run GPU segments on. Requires a valid device id\n");
    printf("\t-help: Prints out available command line parameters, then exits program\n");
    printf("\t-in: Instructs program to include intersections within a single family in the final measure calculation (excluded by default)\n");
    printf("\t-md [int > 0]: Specifies maximum depth of intersections to perform. (Default = number of input sets)\n");
    printf("\t-mt: Instructs code to perform check to see if sets to be intersected are empty or not\n");
    printf("\t-o [int > 0]: MANDATORY!! Specifies the number of features per feature vector\n");
    printf("\t-t: Instructs program to time the code, and print results of the timing\n");
    printf("\t-v: Instructs program to print verbose information while running\n");
}

/******************************************************************************
 * cpuIntersections
 *
 * Sets up the algorithm to perform all finite intersections on the CPU
 *
 * [in]:
 *         intersections: A pointer to an array holding all fundamental subsets,
 *             and space for the output of the intersections
 *
 *         prefixPascal: A pointer to an array of prefix summed Pascal Numbers used
 *             to determine how many intersections to perform
 *
 *         time_code: Boolean determining whether to time the code or not
 * [out]:
 *         intersections: The results of all intersections saved to the array
 *
 * [return]:
 *         void
 *
 *******************************************************************************/
void cpuIntersections(float* intersections, float* prefixPascal, bool time_code){
    pthread_t* threads = new pthread_t[CORES];
    void *status;

    printf("Performing CPU (%i Cores) Power Set on %i Fundamental Subsets\n",CORES, F_SUBSET_COUNT);
    //Time and perform the intersections on the CPU   
    clock_t st = clock();
    
    float curPascal = F_SUBSET_COUNT;
    for(unsigned i = 1; i < maxDepth; ++i){
        cpuDepth++;
        //Get next Pascal number
        curPascal = curPascal * ((F_SUBSET_COUNT - i) / (i + 1.0));
        bitPermute = (1 << i + 1) - 1;
        bitCount = 0;
        
        for(unsigned j = 0; j < CORES; ++j){
            //Prepare the parameters for the intersect CPU thread function
            intersectArgs* args = (intersectArgs*)malloc(sizeof(args));
            args->a = intersections;
            args->pascal = curPascal;
            args->prefixes = prefixPascal;
            //Call intersect function to perform an intersection on 
            // the sets #leastBit and #(bitPattern-leastBit)
            pthread_create(&threads[j], NULL, intersectCPU, args);
        }
        for(unsigned j = 0; j < CORES; ++j){
            pthread_join(threads[j], &status);
        }
    }

    //End timing and print out runtime
    if(time_code == true){
        clock_t ed = clock();
        clock_t stm = clock();
        clock_t edm = clock();
        cout << "Elapsed time on host: "<<(((float)((ed - st) / CORES) + (edm - stm) ) / 
            (float)CLOCKS_PER_SEC) * 1000    << " ms" << std::endl;
    }
}

/******************************************************************************
 * gpuIntersections
 *
 * Sets up and launches the kernels that perform intersections on the GPU
 *
 * [in]:
 *         intersections: A pointer to an array holding all fundamental subsets,
 *             and space for the output of the intersections
 *
 *         prefixPascal: A pointer to an array of prefix summed Pascal Numbers used
 *             to determine how many intersections to perform
 *
 *         time_code: Boolean determining whether to time the code or not
 *
 *         emptySetSize: Determines how many sets will result from the finite intersections
 *
 * [out]:
 *         intersections: The results of all intersections saved to the array
 *
 * [return]:
 *         void
 *
 *******************************************************************************/
void gpuIntersections(float* intersections, float* prefixPascal, bool time_code, 
                          unsigned emptySetSize){
    //A set to track which sets are empty/non-empty,
    float *emptySets = new float[emptySetSize];
    float *deviceEmptySets;
    for(bitString i = 0; i < emptySetSize; ++i){
        emptySets[i] = minFloat;
    }

    if(emptySetCheck == true){
        CUDA_CHECK_RETURN(hipMalloc((void**)&deviceEmptySets, emptySetSize*sizeof(float)));
        CUDA_CHECK_RETURN(hipMemcpy(deviceEmptySets, emptySets, emptySetSize*sizeof(float),
        hipMemcpyHostToDevice));
    }

    printf("Performing GPU Power Set on %i Fundamental Subsets\n", F_SUBSET_COUNT);

    //Declare array to track which sets in intersections set are not empty sets
    //Set up timer code
    hipEvent_t start, stop;
    float elapsedTime;
    hipStream_t* streams = new hipStream_t[STREAMS];
    if(time_code == true){
        CUDA_CHECK_RETURN(hipEventCreate(&start));
        CUDA_CHECK_RETURN(hipEventCreate(&stop));
        CUDA_CHECK_RETURN(hipEventRecord(start, 0));
    }

    float curPascal = F_SUBSET_COUNT;
    for(unsigned j = 0; j < STREAMS; ++j){
        CUDA_CHECK_RETURN(hipStreamCreate(&streams[j])); //Create streams
    }

    for(unsigned i = 1; i < maxDepth; ++i){
        //Use pascal numbers to determine how many intersections are performed at this level
        curPascal = curPascal * ((F_SUBSET_COUNT - i)/ (i + 1.0));
        bitString bitPattern = (1 << i + 1) -1; //Get the first combination of i bits

        if(emptySetCheck == true){
            //For every intersection of i sets
            CUDA_CHECK_RETURN(hipMemcpy(emptySets, deviceEmptySets, emptySetSize*
                                 sizeof(float), hipMemcpyDeviceToHost));
            CUDA_CHECK_RETURN(hipGetLastError());
        }
        for(bitString j = 0; j < curPascal; ++j){
            bitString leastBit = bitPattern & -bitPattern;
            bitString setAIndex = prefixPascal[i] + 
                getBitPatternIndex(i, bitPattern - leastBit);
            //Check if the sets to be intersected are empty
            if(emptySetCheck == false || emptySets[setAIndex] > minFloat || i == 1){
                unsigned myStream = j % STREAMS;
                //Launch intersection into stream j
                intersectKernel <<< 
                                   1 + (VECTORS_PER_SUBSET / TILE_WIDTH), TILE_WIDTH, 
                                   TILE_WIDTH * (VECTOR_SIZE + 1) * sizeof(float) * 2, 
                                   streams[myStream] 
                                >>>
                                (
                                    intersections, F_SUBSET_COUNT, VECTORS_PER_SUBSET, 
                                    VECTOR_SIZE, 1 + getBitPatternIndex(1, leastBit), setAIndex, 
                                    prefixPascal[i+1] + j, minFloat, WIDTH
                                );
                CUDA_CHECK_RETURN(hipGetLastError());

                if(emptySetCheck == true){
                    //Determine if the intersection performed yielded the empty set
                    isEmptyKernel <<< 
                                     (unsigned)ceil((float)VECTORS_PER_SUBSET / 
                                         min(VECTORS_PER_SUBSET, 512)), 
                                     min(VECTORS_PER_SUBSET, 512), 0, streams[myStream] 
                                  >>>
                                  (
                                     intersections, deviceEmptySets, prefixPascal[i+1] + j, 
                                     VECTORS_PER_SUBSET, minFloat
                                  );
                }
            }
            //Get the next combination of bits
            bitPattern = next_perm(bitPattern);
        }
        hipDeviceSynchronize();
    }

    //Destroy all streams
    for(unsigned j = 0; j < STREAMS; ++j){
        CUDA_CHECK_RETURN(hipStreamDestroy(streams[j]));
    }
    hipDeviceSynchronize();

    if(time_code == true){
        CUDA_CHECK_RETURN(hipDeviceSynchronize());// Wait for the GPU launched work to complete

        CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
        CUDA_CHECK_RETURN(hipEventSynchronize(stop));
        CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));

        CUDA_CHECK_RETURN(hipEventDestroy(start));
        CUDA_CHECK_RETURN(hipEventDestroy(stop));
        cout << "Elapsed kernel time: " << elapsedTime << " ms" << std::endl;
    }
    CUDA_CHECK_RETURN(hipGetLastError());
    hipFree(deviceEmptySets);
}

/***
*	START OF ACS-4953 CHANGES
*/

template <typename T>
T* setDifferenceOfFamilies(
	T* familyA,
	T* familyB
) {
	T* setDifferenceResult = new T[SUBSETS_PER_FAMILY * VECTORS_PER_SUBSET * VECTOR_SIZE];
	unsigned* vectorsInCommonCounts = new unsigned[SUBSETS_PER_FAMILY * SUBSETS_PER_FAMILY];

	//initilize counts to 0.  These will be incremented as vectors that match are found
	for (unsigned i = 0; i < SUBSETS_PER_FAMILY * SUBSETS_PER_FAMILY; i++) {
		vectorsInCommonCounts[i] = 0;
	}

	//find vectors in common
	//for each vector in A
	for (unsigned i = 0; i < SUBSETS_PER_FAMILY * VECTORS_PER_SUBSET; i++) {
		//get the subset index to index into vectorsInCommonCounts
		unsigned vectorInASubsetIndex = floorf((float)i / VECTORS_PER_SUBSET);
		//for each vector in B
		for (unsigned j = 0; j < SUBSETS_PER_FAMILY * VECTORS_PER_SUBSET; j++) {
			//get the subset index to index into vectorsInCommonCounts
			unsigned vectorInBSubsetIndex = floorf((float)j / VECTORS_PER_SUBSET);
			bool vectorsMatch = true;
			for (unsigned k = 0; vectorsMatch && k < VECTOR_SIZE; k++) {
				if (familyA[i * VECTOR_SIZE + k] != familyB[j * VECTOR_SIZE + k]) {
					vectorsMatch = false;
				}
			}
			if (vectorsMatch) {
				vectorsInCommonCounts[vectorInASubsetIndex * SUBSETS_PER_FAMILY + vectorInBSubsetIndex]++;
			}
		}
	}

	//write to output array
	//for each subset in A
	for (unsigned i = 0; i < SUBSETS_PER_FAMILY; i++) {
		//if the vectorsInCommonCounts of any element in the ith row is VECTOR_SIZE...
		bool subsetsMatch = false;
		for (unsigned j = 0; !subsetsMatch && j < SUBSETS_PER_FAMILY; j++) {
			if (vectorsInCommonCounts[i * SUBSETS_PER_FAMILY + j] == VECTORS_PER_SUBSET) {
				subsetsMatch = true;
			}
		}
		//write each term of the subset as minFloat.  Otherwise, preserve the value
		for (unsigned j = 0; j < VECTORS_PER_SUBSET * VECTOR_SIZE; j++) {
			if (subsetsMatch) {
				setDifferenceResult[i * VECTORS_PER_SUBSET * VECTOR_SIZE + j] = minFloat;
			}
			else {
				setDifferenceResult[i * VECTORS_PER_SUBSET * VECTOR_SIZE + j] =
					familyA[i * VECTORS_PER_SUBSET * VECTOR_SIZE + j];
			}
		}
	}

	return setDifferenceResult;
}

unsigned getFamilyCardinality(float* input, unsigned size) {
	unsigned setSize = F_SUBSET_COUNT / 2;
	unsigned index = 0;
	//for each subset in input family of sets
	while (index < setSize) {
		//if we encounter a subset with a vector that starts with minFloat
		//swap the subsets with the row-major index of our final subset based on our running setSize
		//decrease the set size if this is the case (we have a 'nulled'
		//out subset from set difference on the families)
		if (input[index * VECTOR_SIZE * VECTORS_PER_SUBSET] == minFloat) {
			for (unsigned i = 0; i < VECTOR_SIZE * VECTORS_PER_SUBSET; i++) {
				float temp = input[(index * VECTOR_SIZE * VECTORS_PER_SUBSET) + i];
				input[(index * VECTOR_SIZE * VECTORS_PER_SUBSET) + i] =
					input[(setSize * VECTOR_SIZE * VECTORS_PER_SUBSET) -
					((VECTOR_SIZE * VECTORS_PER_SUBSET) - i)];
				input[(setSize * VECTOR_SIZE * VECTORS_PER_SUBSET) -
					((VECTOR_SIZE * VECTORS_PER_SUBSET) - i)] = temp;
			}
			setSize--;
		}
		else {
			index++;
		}
	}
	return setSize;
}

template <typename T>
__global__ void descriptiveIntersectionGPU(
	T* d_A,
	T* d_B,
	unsigned* d_freqA,
	unsigned* d_freqB,
	T* d_output,
	float minFloat,
	unsigned SUBSETS_PER_FAMILY,
	unsigned VECTORS_PER_SUBSET,
	unsigned VECTOR_SIZE,
	float tolerance
) {

	extern __shared__ T shared[];

	T* ds_A = &shared[0];


	unsigned vectorInFamily = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned setSubscript = floorf((float)vectorInFamily / VECTORS_PER_SUBSET);
	int numberOfVectorsToLoad = SUBSETS_PER_FAMILY * VECTORS_PER_SUBSET;

	//Load A into shared memory
	for (unsigned i = 0; i < VECTOR_SIZE; i++) {
		if (vectorInFamily < SUBSETS_PER_FAMILY * VECTORS_PER_SUBSET) {
			ds_A[threadIdx.x * VECTOR_SIZE + i] = d_A[vectorInFamily * VECTOR_SIZE + i];
		}
	}

	__syncthreads();

	//Get subset descriptions before intersecting
	if (vectorInFamily < SUBSETS_PER_FAMILY * VECTORS_PER_SUBSET) {
		//get vector frequencies (minFloats will be 0)
		if (ds_A[threadIdx.x * VECTOR_SIZE] != minFloat) {
			for (unsigned i = 0; i < VECTORS_PER_SUBSET; i++) {
				bool vectorsMatch = true;
				for (unsigned j = 0; vectorsMatch && j < VECTOR_SIZE; j++) {
					if (ds_A[threadIdx.x * VECTOR_SIZE + j] !=
						d_A[(setSubscript * VECTORS_PER_SUBSET * VECTOR_SIZE) + (i * VECTOR_SIZE) + j]) {
						vectorsMatch = false;
					}
				}
				//every vector should match with itself at least, making the freq 1
				if (vectorsMatch) {
					d_freqA[vectorInFamily]++;
				}
			}
		}

		if (d_B[vectorInFamily * VECTOR_SIZE] != minFloat) {
			for (unsigned i = 0; i < VECTORS_PER_SUBSET; i++) {
				bool vectorsMatch = true;
				for (unsigned j = 0; vectorsMatch && j < VECTOR_SIZE; j++) {
					if (d_B[vectorInFamily * VECTOR_SIZE + j] !=
						d_B[(setSubscript * VECTORS_PER_SUBSET * VECTOR_SIZE) + (i * VECTOR_SIZE) + j]) {
						vectorsMatch = false;
					}
				}
				//every vector should match with itself at least, making the freq 1
				if (vectorsMatch) {
					d_freqB[vectorInFamily]++;
				}
			}
		}
	}
	__syncthreads();

	if (vectorInFamily < SUBSETS_PER_FAMILY * VECTORS_PER_SUBSET) {
		//handle if frequencies greater than 1, all else will be left as is
		bool threadhandlingRepeatedVectorInA = false;
		if (d_freqA[vectorInFamily] > 1) {
			//find first occurance of repeated vector
			for (unsigned i = 0; !threadhandlingRepeatedVectorInA && i < VECTORS_PER_SUBSET; i++) {
				bool vectorsMatch = true;
				for (unsigned j = 0; vectorsMatch && j < VECTOR_SIZE; j++) {
					if (ds_A[threadIdx.x * VECTOR_SIZE + j] !=
						d_A[(setSubscript * VECTORS_PER_SUBSET * VECTOR_SIZE) + (i * VECTOR_SIZE) + j]) {
						vectorsMatch = false;
					}
				}
				if (vectorsMatch) {
					if (vectorInFamily * VECTOR_SIZE >
						(setSubscript * VECTORS_PER_SUBSET * VECTOR_SIZE) + (i * VECTOR_SIZE)) {
						threadhandlingRepeatedVectorInA = true;
					}
				}
			}
		}

		//overwrite repeated vectors in A with minFloats
		if (threadhandlingRepeatedVectorInA) {
			for (unsigned i = 0; i < VECTOR_SIZE; i++) {
				//We need to change both since some metrics will use the descriptions after performing intersections
				ds_A[threadIdx.x * VECTOR_SIZE + i] = minFloat;
				d_A[vectorInFamily * VECTOR_SIZE + i] = minFloat;
			}
		}

		bool threadhandlingRepeatedVectorInB = false;
		if (d_freqB[vectorInFamily] > 1) {
			//find first occurance of repeated vector
			for (unsigned i = 0; !threadhandlingRepeatedVectorInB && i < VECTORS_PER_SUBSET; i++) {
				bool vectorsMatch = true;
				for (unsigned j = 0; vectorsMatch && j < VECTOR_SIZE; j++) {
					if (d_B[vectorInFamily * VECTOR_SIZE + j] !=
						d_B[(setSubscript * VECTORS_PER_SUBSET * VECTOR_SIZE) + (i * VECTOR_SIZE) + j]) {
						vectorsMatch = false;
					}
				}
				if (vectorsMatch) {
					if (vectorInFamily * VECTOR_SIZE >
						(setSubscript * VECTORS_PER_SUBSET * VECTOR_SIZE) + (i * VECTOR_SIZE)) {
						threadhandlingRepeatedVectorInB = true;
					}
				}
			}
		}

		//overwrite repeated vectors in B with minFloats
		if (threadhandlingRepeatedVectorInB) {
			for (unsigned i = 0; i < VECTOR_SIZE; i++) {
				d_B[vectorInFamily * VECTOR_SIZE + i] = minFloat;
			}
		}
	}
	__syncthreads();

	//Perform Intersections
	//for each subset in B...
	if (vectorInFamily < SUBSETS_PER_FAMILY * VECTORS_PER_SUBSET) {
		for (unsigned i = 0; i < SUBSETS_PER_FAMILY; i++) {
			//for each vector in subset of B...
			bool vectorIsInSubset = false;
			for (unsigned j = 0; !vectorIsInSubset && j < VECTORS_PER_SUBSET; j++) {
				bool vectorsMatch = true;
				for (unsigned k = 0; vectorsMatch && k < VECTOR_SIZE; k++) {
					if (abs(d_B[i * VECTORS_PER_SUBSET * VECTOR_SIZE + j * VECTOR_SIZE + k] -//!=
						ds_A[threadIdx.x * VECTOR_SIZE + k]) > tolerance) {
						vectorsMatch = false;
					}
				}
				//if the vector is found within subset, don't check the rest of the subset
				if (vectorsMatch) {
					vectorIsInSubset = true;
				}
			}
			for (unsigned j = 0; j < VECTOR_SIZE; j++) {
				if (vectorIsInSubset) {
					d_output[(i * VECTOR_SIZE * VECTORS_PER_SUBSET) + (vectorInFamily * VECTOR_SIZE) +
						(setSubscript * (SUBSETS_PER_FAMILY - 1) * VECTOR_SIZE * VECTORS_PER_SUBSET) + j] =
						ds_A[threadIdx.x * VECTOR_SIZE + j];
				}
				else {
					d_output[(i * VECTOR_SIZE * VECTORS_PER_SUBSET) + (vectorInFamily * VECTOR_SIZE) +
						(setSubscript * (SUBSETS_PER_FAMILY - 1) * VECTOR_SIZE * VECTORS_PER_SUBSET) + j] =
						minFloat;
				}
			}
		}
	}
}

template <typename T>
__global__ void runMetricOnGPU(
	pseudometric_t<T> pseudometric,
	T* d_A,
	T* d_B,
	T* d_inter,
	T* result,
	unsigned sizeOfA,
	unsigned sizeOfB,
	float minFloat,
	unsigned VECTOR_SIZE,
	unsigned VECTORS_PER_SUBSET,
	unsigned SUBSETS_PER_FAMILY,
	metric_t<T> embeddedMetric
) {
	unsigned row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned col = blockIdx.x * blockDim.x + threadIdx.x;

	unsigned size = VECTOR_SIZE * VECTORS_PER_SUBSET;

	if (row < sizeOfA && col < sizeOfB) {
		result[row * sizeOfB + col] = (*pseudometric)(
			d_A,
			d_B,
			d_inter,
			row,
			col,
			size,
			minFloat,
			VECTOR_SIZE,
			VECTORS_PER_SUBSET,
			SUBSETS_PER_FAMILY,
			embeddedMetric
			);
	}
	else {
		result[row * sizeOfB + col] = 0;
	}
}

//Version of d-iterated pseudometric that uses GPU for metric caluculations and descriptive intersections
template <typename T>
T dIteratedPseudometricGPU(
	T* family_A,
	T* family_B,
	bool time_code,
	pseudometric_t<T>* pseudometric,
	metric_t<T>* embeddedMetric = &p_no_embeddedMetric<T>,
	float tolerance = 0.0f
) {
	//Device Variables
	pseudometric_t<T> d_pseudometric;
	metric_t<T> d_metric;
	T* d_A;
	T* d_B;
	T* d_inter;
	T* d_family_A_less_B;
	T* d_family_B_less_A;
	unsigned* d_freqA;
	unsigned* d_freqB;

	//Host Variables
	unsigned sizeOfFamilyAUnionFamilyB;
	bool familiesAreDisjoint = true;
	unsigned numberOfVectorsPerFamily = SUBSETS_PER_FAMILY * VECTORS_PER_SUBSET;
	unsigned intersectionSize = pow(SUBSETS_PER_FAMILY, 2) * VECTORS_PER_SUBSET * VECTOR_SIZE;
	unsigned indiciesPerFamily = VECTORS_PER_SUBSET * VECTOR_SIZE * SUBSETS_PER_FAMILY;
	unsigned subsetSize = VECTOR_SIZE * VECTORS_PER_SUBSET;
	T* h_inter = new T[intersectionSize];
	T result = 0.0;
	T* h_family_A_less_B = setDifferenceOfFamilies(family_A, family_B);
	T* h_family_B_less_A = setDifferenceOfFamilies(family_B, family_A);
	unsigned sizeOfFamilyALessB = getFamilyCardinality(h_family_A_less_B, indiciesPerFamily);
	unsigned sizeOfFamilyBLessA = getFamilyCardinality(h_family_B_less_A, indiciesPerFamily);

	unsigned* h_freqA = new unsigned[numberOfVectorsPerFamily];
	unsigned* h_freqB = new unsigned[numberOfVectorsPerFamily];
	for (unsigned i = 0; i < numberOfVectorsPerFamily; i++) {
		h_freqA[i] = 0;
		h_freqB[i] = 0;
	}

	if (sizeOfFamilyALessB == SUBSETS_PER_FAMILY && sizeOfFamilyBLessA == SUBSETS_PER_FAMILY) {
		//If the families A and B are disjoint, then the cardinality of their union 
		//is the sum of their cardinalities
		sizeOfFamilyAUnionFamilyB = 2 * SUBSETS_PER_FAMILY;
	}
	else {
		//Otherwise, take the cardinality of B, and sum it with the cardinality of A less B 
		sizeOfFamilyAUnionFamilyB = SUBSETS_PER_FAMILY + sizeOfFamilyALessB;
		familiesAreDisjoint = false;
	}

	//allocate to device
	hipMalloc((void**)&d_A, sizeof(T) * indiciesPerFamily);
	hipMalloc((void**)&d_B, sizeof(T) * indiciesPerFamily);
	hipMalloc((void**)&d_family_A_less_B, sizeof(T) * indiciesPerFamily);
	hipMalloc((void**)&d_family_B_less_A, sizeof(T) * indiciesPerFamily);
	hipMalloc((void**)&d_inter, sizeof(T) * intersectionSize);
	hipMalloc((void**)&d_freqA, sizeof(unsigned) * numberOfVectorsPerFamily);
	hipMalloc((void**)&d_freqB, sizeof(unsigned) * numberOfVectorsPerFamily);

	//copy to device
	hipMemcpy(d_A, family_A, sizeof(T) * indiciesPerFamily, hipMemcpyHostToDevice);
	hipMemcpy(d_B, family_B, sizeof(T) * indiciesPerFamily, hipMemcpyHostToDevice);
	hipMemcpy(d_family_A_less_B, h_family_A_less_B, sizeof(T) * indiciesPerFamily, hipMemcpyHostToDevice);
	hipMemcpy(d_family_B_less_A, h_family_B_less_A, sizeof(T) * indiciesPerFamily, hipMemcpyHostToDevice);
	hipMemcpy(d_freqA, h_freqA, sizeof(unsigned) * numberOfVectorsPerFamily, hipMemcpyHostToDevice);
	hipMemcpy(d_freqB, h_freqB, sizeof(unsigned) * numberOfVectorsPerFamily, hipMemcpyHostToDevice);

	T* d_result;
	T* h_result = new T[(SUBSETS_PER_FAMILY * SUBSETS_PER_FAMILY)];
	hipMalloc(&d_result, sizeof(T) * (SUBSETS_PER_FAMILY * SUBSETS_PER_FAMILY));

	// Copy device function pointer to host side
	hipMemcpyFromSymbol(&d_pseudometric, HIP_SYMBOL(*pseudometric), sizeof(pseudometric_t<T>));
	hipMemcpyFromSymbol(&d_metric, HIP_SYMBOL(*embeddedMetric), sizeof(metric_t<T>));

	//play with this to get better results (use with kernel timing) ... biggest block size my card can handle
	unsigned TILE_WIDTH_METRIC = 16;

	dim3 metricGrid(
		ceil((double)SUBSETS_PER_FAMILY / TILE_WIDTH_METRIC),
		ceil((double)SUBSETS_PER_FAMILY / TILE_WIDTH_METRIC),
		1
	);
	dim3 metricBlock(TILE_WIDTH_METRIC, TILE_WIDTH_METRIC, 1);

	dim3 intersectionGrid(ceil((double)numberOfVectorsPerFamily / TILE_WIDTH), 1, 1);
	dim3 intersectionBlock(TILE_WIDTH, 1, 1);

	hipEvent_t start, stop;
	float elapsedTime;
	if (time_code == true) {
		CUDA_CHECK_RETURN(hipEventCreate(&start));
		CUDA_CHECK_RETURN(hipEventCreate(&stop));
		CUDA_CHECK_RETURN(hipEventRecord(start, 0));
	}

	descriptiveIntersectionGPU<T> << <intersectionGrid, intersectionBlock, VECTOR_SIZE* TILE_WIDTH * sizeof(T) >> > (
		d_A,
		d_family_B_less_A,
		d_freqA,
		d_freqB,
		d_inter,
		minFloat,
		SUBSETS_PER_FAMILY,
		VECTORS_PER_SUBSET,
		VECTOR_SIZE,
		tolerance
		);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	if (time_code) {
		CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
		CUDA_CHECK_RETURN(hipEventSynchronize(stop));
		CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));

		CUDA_CHECK_RETURN(hipEventDestroy(start));
		CUDA_CHECK_RETURN(hipEventDestroy(stop));
		if (familiesAreDisjoint) {
			cout << "Elapsed kernel time for intersections of elements of A and B (A and B are disjoint): " << elapsedTime << " ms" << std::endl;
		}
		else {
			cout << "Elapsed kernel time for intersections of elements of A and B - A: " << elapsedTime << " ms" << std::endl;
		}
	}
	CUDA_CHECK_RETURN(hipGetLastError());

	if (time_code) {
		CUDA_CHECK_RETURN(hipEventCreate(&start));
		CUDA_CHECK_RETURN(hipEventCreate(&stop));
		CUDA_CHECK_RETURN(hipEventRecord(start, 0));
	}

	runMetricOnGPU<T> << <metricGrid, metricBlock >> > (
		d_pseudometric,
		d_A,
		d_family_B_less_A,
		d_inter,
		d_result,
		SUBSETS_PER_FAMILY,
		sizeOfFamilyBLessA,
		minFloat,
		VECTOR_SIZE,
		VECTORS_PER_SUBSET,
		SUBSETS_PER_FAMILY,
		d_metric
		);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	if (time_code) {
		CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
		CUDA_CHECK_RETURN(hipEventSynchronize(stop));
		CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));

		CUDA_CHECK_RETURN(hipEventDestroy(start));
		CUDA_CHECK_RETURN(hipEventDestroy(stop));
		cout << "Elapsed kernel time for running metrics: " << elapsedTime << " ms" << std::endl;
	}
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy(h_result, d_result, sizeof(T) * (SUBSETS_PER_FAMILY * SUBSETS_PER_FAMILY),
		hipMemcpyDeviceToHost));

	T result1 = 0;
	for (unsigned i = 0; i < (SUBSETS_PER_FAMILY * SUBSETS_PER_FAMILY); i++) {
		result1 += h_result[i];
	}

	result1 /= (SUBSETS_PER_FAMILY * sizeOfFamilyAUnionFamilyB);

	if (!familiesAreDisjoint) {

		//reset frequency counts
		hipMemcpy(d_freqA, h_freqA, sizeof(unsigned) * numberOfVectorsPerFamily, hipMemcpyHostToDevice);
		hipMemcpy(d_freqB, h_freqB, sizeof(unsigned) * numberOfVectorsPerFamily, hipMemcpyHostToDevice);
		
		if(time_code) {
			CUDA_CHECK_RETURN(hipEventCreate(&start));
			CUDA_CHECK_RETURN(hipEventCreate(&stop));
			CUDA_CHECK_RETURN(hipEventRecord(start, 0));
		}

		descriptiveIntersectionGPU<T> << <intersectionGrid, intersectionBlock, VECTOR_SIZE* TILE_WIDTH * sizeof(T) >> > (
			d_family_A_less_B,
			d_B,
			d_freqA,
			d_freqB,
			d_inter,
			minFloat,
			SUBSETS_PER_FAMILY,
			VECTORS_PER_SUBSET,
			VECTOR_SIZE,
			tolerance
			);

		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		if (time_code) {
			CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
			CUDA_CHECK_RETURN(hipEventSynchronize(stop));
			CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));

			CUDA_CHECK_RETURN(hipEventDestroy(start));
			CUDA_CHECK_RETURN(hipEventDestroy(stop));
			cout << "Elapsed kernel time for intersections of elements of A - B and B: " << elapsedTime << " ms" << std::endl;
		}
		CUDA_CHECK_RETURN(hipGetLastError());

		if (time_code) {
			CUDA_CHECK_RETURN(hipEventCreate(&start));
			CUDA_CHECK_RETURN(hipEventCreate(&stop));
			CUDA_CHECK_RETURN(hipEventRecord(start, 0));
		}

		runMetricOnGPU<T> << <metricGrid, metricBlock >> > (
			d_pseudometric,
			d_family_A_less_B,
			d_B,
			d_inter,
			d_result,
			sizeOfFamilyALessB,
			SUBSETS_PER_FAMILY,
			minFloat,
			VECTOR_SIZE,
			VECTORS_PER_SUBSET,
			SUBSETS_PER_FAMILY,
			d_metric
			);

		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		if (time_code) {
			CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
			CUDA_CHECK_RETURN(hipEventSynchronize(stop));
			CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));

			CUDA_CHECK_RETURN(hipEventDestroy(start));
			CUDA_CHECK_RETURN(hipEventDestroy(stop));
			cout << "Elapsed kernel time for running metrics: " << elapsedTime << " ms" << std::endl;
		}
		CUDA_CHECK_RETURN(hipGetLastError());
		CUDA_CHECK_RETURN(hipMemcpy(h_result, d_result, sizeof(T) * (SUBSETS_PER_FAMILY * SUBSETS_PER_FAMILY),
			hipMemcpyDeviceToHost));

		T result2 = 0;
		for (unsigned i = 0; i < (SUBSETS_PER_FAMILY * SUBSETS_PER_FAMILY); i++) {
			result2 += h_result[i];
		}
		result2 /= (SUBSETS_PER_FAMILY * sizeOfFamilyAUnionFamilyB);

		result = result1 + result2;
	}
	else {
		result = result1 * 2;
	}

	CUDA_CHECK_RETURN(hipFree((void*)d_A));
	CUDA_CHECK_RETURN(hipFree((void*)d_B));
	CUDA_CHECK_RETURN(hipFree((void*)d_freqA));
	CUDA_CHECK_RETURN(hipFree((void*)d_freqB));
	CUDA_CHECK_RETURN(hipFree((void*)d_inter));
	CUDA_CHECK_RETURN(hipFree((void*)d_family_A_less_B));
	CUDA_CHECK_RETURN(hipFree((void*)d_family_B_less_A));
	CUDA_CHECK_RETURN(hipFree((void*)d_result));
	CUDA_CHECK_RETURN(hipDeviceReset());
	return result;
}

template <typename T>
T* runMetricOnCPU(
	pseudometric_t<T> pseudometric,
	T* desc_A,
	T* desc_B,
	T* desc_inter,
	unsigned sizeOfA,
	unsigned sizeOfB,
	metric_t<T> embeddedMetric
) {
	unsigned size = VECTOR_SIZE * VECTORS_PER_SUBSET;
	if (sizeOfA == 0 || sizeOfB == 0)
		return 0;
	T* result = new T[sizeOfA * sizeOfB];
	for (unsigned i = 0; i < sizeOfA; i++) {
		for (unsigned j = 0; j < sizeOfB; j++) {
			result[i * sizeOfB + j] = (*pseudometric)(
				desc_A,
				desc_B,
				desc_inter,
				i,
				j,
				size,
				minFloat,
				VECTOR_SIZE,
				VECTORS_PER_SUBSET,
				SUBSETS_PER_FAMILY,
				embeddedMetric
				);
		}
	}

	return result;
}

//Version of d-iterated pseudometric that uses GPU for descriptive intersections
template <typename T>
T dIteratedPseudometric(
	T* family_A,
	T* family_B,
	bool time_code,
	pseudometric_t<T> pseudometric,
	metric_t<T> embeddedMetric = p_no_embeddedMetric<T>,
	float tolerance = 0.0f
) {
	//Device Variables
	T* d_A;
	T* d_B;
	T* d_inter;
	T* d_family_A_less_B;
	T* d_family_B_less_A;
	unsigned* d_freqA;
	unsigned* d_freqB;

	//Host Variables
	unsigned sizeOfFamilyAUnionFamilyB;
	bool familiesAreDisjoint = true;
	unsigned numberOfVectorsPerFamily = SUBSETS_PER_FAMILY * VECTORS_PER_SUBSET;
	unsigned intersectionSize = pow(SUBSETS_PER_FAMILY, 2) * VECTORS_PER_SUBSET * VECTOR_SIZE;
	unsigned indiciesPerFamily = VECTORS_PER_SUBSET * VECTOR_SIZE * SUBSETS_PER_FAMILY;
	unsigned subsetSize = VECTOR_SIZE * VECTORS_PER_SUBSET;
	T* h_inter = new T[intersectionSize];
	T result = 0.0;
	T* h_family_A_less_B = setDifferenceOfFamilies(family_A, family_B);
	T* h_family_B_less_A = setDifferenceOfFamilies(family_B, family_A);
	unsigned sizeOfFamilyALessB = getFamilyCardinality(h_family_A_less_B, indiciesPerFamily);
	unsigned sizeOfFamilyBLessA = getFamilyCardinality(h_family_B_less_A, indiciesPerFamily);

	unsigned* h_freqA = new unsigned[numberOfVectorsPerFamily];
	unsigned* h_freqB = new unsigned[numberOfVectorsPerFamily];
	for (unsigned i = 0; i < numberOfVectorsPerFamily; i++) {
		h_freqA[i] = 0;
		h_freqB[i] = 0;
	}

	if (sizeOfFamilyALessB == SUBSETS_PER_FAMILY && sizeOfFamilyBLessA == SUBSETS_PER_FAMILY) {
		//If the families A and B are disjoint, then the cardinality of their union 
		//is the sum of their cardinalities
		sizeOfFamilyAUnionFamilyB = 2 * SUBSETS_PER_FAMILY;
	}
	else {
		//Otherwise, take the cardinality of B, and sum it with the cardinality of A less B 
		sizeOfFamilyAUnionFamilyB = SUBSETS_PER_FAMILY + sizeOfFamilyALessB;
		familiesAreDisjoint = false;
	}

	//allocate to device
	hipMalloc((void**)&d_A, sizeof(T) * indiciesPerFamily);
	hipMalloc((void**)&d_B, sizeof(T) * indiciesPerFamily);
	hipMalloc((void**)&d_family_A_less_B, sizeof(T) * indiciesPerFamily);
	hipMalloc((void**)&d_family_B_less_A, sizeof(T) * indiciesPerFamily);
	hipMalloc((void**)&d_inter, sizeof(T) * intersectionSize);
	hipMalloc((void**)&d_freqA, sizeof(unsigned) * numberOfVectorsPerFamily);
	hipMalloc((void**)&d_freqB, sizeof(unsigned) * numberOfVectorsPerFamily);

	//copy to device
	hipMemcpy(d_A, family_A, sizeof(T) * indiciesPerFamily, hipMemcpyHostToDevice);
	hipMemcpy(d_B, family_B, sizeof(T) * indiciesPerFamily, hipMemcpyHostToDevice);
	hipMemcpy(d_family_A_less_B, h_family_A_less_B, sizeof(T) * indiciesPerFamily, hipMemcpyHostToDevice);
	hipMemcpy(d_family_B_less_A, h_family_B_less_A, sizeof(T) * indiciesPerFamily, hipMemcpyHostToDevice);
	hipMemcpy(d_freqA, h_freqA, sizeof(unsigned) * numberOfVectorsPerFamily, hipMemcpyHostToDevice);
	hipMemcpy(d_freqB, h_freqB, sizeof(unsigned) * numberOfVectorsPerFamily, hipMemcpyHostToDevice);

	dim3 intersectionGrid(ceil((double)numberOfVectorsPerFamily / TILE_WIDTH), 1, 1);
	dim3 intersectionBlock(TILE_WIDTH, 1, 1);

	hipEvent_t start, stop;
	float elapsedTime;
	if (time_code) {
		CUDA_CHECK_RETURN(hipEventCreate(&start));
		CUDA_CHECK_RETURN(hipEventCreate(&stop));
		CUDA_CHECK_RETURN(hipEventRecord(start, 0));
	}

	descriptiveIntersectionGPU<T> << <intersectionGrid, intersectionBlock, VECTOR_SIZE* TILE_WIDTH * sizeof(T) >> > (
		d_A,
		d_family_B_less_A,
		d_freqA,
		d_freqB,
		d_inter,
		minFloat,
		SUBSETS_PER_FAMILY,
		VECTORS_PER_SUBSET,
		VECTOR_SIZE,
		tolerance
		);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	if (time_code) {
		CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
		CUDA_CHECK_RETURN(hipEventSynchronize(stop));
		CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));

		CUDA_CHECK_RETURN(hipEventDestroy(start));
		CUDA_CHECK_RETURN(hipEventDestroy(stop));
		if (familiesAreDisjoint) {
			cout << "Elapsed kernel time for intersections of elements of A and B (A and B are disjoint): " << elapsedTime << " ms" << std::endl;
		} else {
			cout << "Elapsed kernel time for intersections of elements of A and B - A: " << elapsedTime << " ms" << std::endl;
		}
	}
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy(h_inter, d_inter, sizeof(T) * intersectionSize, hipMemcpyDeviceToHost));
	if (PSEUDOMETRIC_USES_DESCRIPTIVE_INTERSECTIONS) {
		CUDA_CHECK_RETURN(hipMemcpy(family_A, d_A, sizeof(T) * indiciesPerFamily, hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(h_family_B_less_A, d_family_B_less_A, sizeof(T) * indiciesPerFamily, hipMemcpyDeviceToHost));
	}

	clock_t st = clock();

	T* metricValues1 = runMetricOnCPU<T>(
		pseudometric,
		family_A,
		h_family_B_less_A,
		h_inter,
		SUBSETS_PER_FAMILY,
		sizeOfFamilyBLessA,
		embeddedMetric
		);

	if (time_code) {
		clock_t ed = clock();
		clock_t stm = clock();
		clock_t edm = clock();
		cout << "Elapsed time for running metrics on host: " << ((float)((ed - st) + (edm - stm)) /
			(float)CLOCKS_PER_SEC) * 1000 << " ms" << std::endl;
	}

	T result1 = 0;
	for (unsigned i = 0; i < (SUBSETS_PER_FAMILY * sizeOfFamilyBLessA); i++) {
		result1 += metricValues1[i];
	}

	result1 /= (SUBSETS_PER_FAMILY * sizeOfFamilyAUnionFamilyB);

	if (!familiesAreDisjoint) {

		//reset frequency counts
		hipMemcpy(d_freqA, h_freqA, sizeof(unsigned) * numberOfVectorsPerFamily, hipMemcpyHostToDevice);
		hipMemcpy(d_freqB, h_freqB, sizeof(unsigned) * numberOfVectorsPerFamily, hipMemcpyHostToDevice);

		if (time_code) {
			CUDA_CHECK_RETURN(hipEventCreate(&start));
			CUDA_CHECK_RETURN(hipEventCreate(&stop));
			CUDA_CHECK_RETURN(hipEventRecord(start, 0));
		}

		descriptiveIntersectionGPU<T> << <intersectionGrid, intersectionBlock, VECTOR_SIZE * TILE_WIDTH * sizeof(T) >> > (
			d_family_A_less_B,
			d_B,
			d_freqA,
			d_freqB,
			d_inter,
			minFloat,
			SUBSETS_PER_FAMILY,
			VECTORS_PER_SUBSET,
			VECTOR_SIZE,
			tolerance
			);

		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		if (time_code) {
			CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
			CUDA_CHECK_RETURN(hipEventSynchronize(stop));
			CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));

			CUDA_CHECK_RETURN(hipEventDestroy(start));
			CUDA_CHECK_RETURN(hipEventDestroy(stop));
			cout << "Elapsed kernel time for intersections of elements of A - B and B: " << elapsedTime << " ms" << std::endl;
		}
		CUDA_CHECK_RETURN(hipGetLastError());
		CUDA_CHECK_RETURN(hipMemcpy(h_inter, d_inter, sizeof(T) * intersectionSize, hipMemcpyDeviceToHost));
		if (PSEUDOMETRIC_USES_DESCRIPTIVE_INTERSECTIONS) {
			CUDA_CHECK_RETURN(hipMemcpy(family_B, d_B, sizeof(T) * indiciesPerFamily, hipMemcpyDeviceToHost));
			CUDA_CHECK_RETURN(hipMemcpy(h_family_A_less_B, d_family_A_less_B, sizeof(T) * indiciesPerFamily, hipMemcpyDeviceToHost));
		}
		st = clock();

		T* metricValues2 = runMetricOnCPU<T>(
			pseudometric,
			h_family_A_less_B,
			family_B,
			h_inter,
			sizeOfFamilyALessB,
			SUBSETS_PER_FAMILY,
			embeddedMetric
			);

		if (time_code) {
			clock_t ed = clock();
			clock_t stm = clock();
			clock_t edm = clock();
			cout << "Elapsed time for running metrics on host: " << ((float)((ed - st) + (edm - stm)) /
				(float)CLOCKS_PER_SEC) * 1000 << " ms" << std::endl;
		}

		T result2 = 0;
		for (unsigned i = 0; i < (sizeOfFamilyALessB * SUBSETS_PER_FAMILY); i++) {
			result2 += metricValues2[i];
		}
		result2 /= (SUBSETS_PER_FAMILY * sizeOfFamilyAUnionFamilyB);

		result = result1 + result2;
	} else {
		result = result1 * 2;
	}

	CUDA_CHECK_RETURN(hipFree((void*)d_A));
	CUDA_CHECK_RETURN(hipFree((void*)d_B));
	CUDA_CHECK_RETURN(hipFree((void*)d_freqA));
	CUDA_CHECK_RETURN(hipFree((void*)d_freqB));
	CUDA_CHECK_RETURN(hipFree((void*)d_inter));
	CUDA_CHECK_RETURN(hipFree((void*)d_family_A_less_B));
	CUDA_CHECK_RETURN(hipFree((void*)d_family_B_less_A));
	CUDA_CHECK_RETURN(hipDeviceReset());
	return result;
}

/******************************************************************************
 * DeTopS main
 *
 * F_SUBSET_COUNT: The specified number of fundamental subsets the data is to be divided into
 * VECTOR_SIZE:    The specified number of elements each feature vector contains
 * VECTORS_PER_SUBSET: The specified number of feature vectors in a fundamental subset
 *
 * totalSize: The total number of elements in the input data
 * fundamentalSubset: A float array that holds the input data
 * intersections: The descriptions of the fundamental subsets, and all descriptive intersections 
 *                between them. In the case a set description's size < the set's size, the extra 
 *                space for that subset is filled with minimum float values
 *
 * [Command Line parameters]:
 *         discretize_input:
 *             Determines whether the input data will be discretized
 *             Requires -b if used
 *             Default: False
 *             Set true by command param -d
 *
 *         num_bins:
 *             Determines how many discrete values are to be used when discretizing
 *             Default: 3
 *             Set by command param -b [int>0]
 *
 *         inputFile:
 *             Determines which files the data is to be read from
 *             !Must be the last parameter entered, followed only by the input files!
 *             Use either this or -fd
 *             Default: None
 *             Set by command param -f [file1 file2 ... fileN]
 *
 *        useCPU:
 *            Determine whether the intersections will be performed on CPU or GPU
 *            Default: False (Run intersections on GPU)
 *            Set true by command param
 *                -c (Sets useGPU false) or
 *                -cg (Sets useCPU and useGPU true)
 *
 *        emptySetCheck:
 *            Determine whether the GPU will check if a set is empty before performing intersection
 *            Setting true may speed up or slow down results, 
 *                 depending on the data, but the output will be the same
 *            Default: False
 *            Set True by command param -mt
 *
 *        time_code:
 *            Determines whether the program will be timed while running or not
 *            Default: False
 *            Set true by command param -t
 *
 *        VECTOR_SIZE:
 *            Determines the number of elements in each feature vector
 *            !Mandatory!
 *            Must be a whole number > 0
 *            Set by command param -o [int>0]
 *
 *        CORES:
 *            Determines how many cores the cpu will run on
 *            Default: 1
 *            Set by command param -cores [int>0]
 *
 *        verbose_info:
 *            Determines if detailed output will be printed
 *            This includes:
 *                Measure for each intersection
 *                Device information
 *                Number of unique feature vectors in each fundamental subset
 *            Default: False
 *            Set true by command param -v
 *        
 *        measure_within_set:
 *            Specify weather measure should be calculated for intersections of sets from one family
 *            Default: False (do not include these in the measure)
 *            Set true by command param -in
 *              
 *        device:
 *            Determines which device the GPU code will run on
 *            Takes in the integer id of a CUDA device
 *            Default: 0
 *            Set by command param -gpu [int>0]
 *
 *        set1, set2:
 *            Determines where to read files from
 *            Takes in an integer followed by two strings
 *            Integer is how many sets to read
 *            String 1 is the file path and name of first set, minus the number
 *            String 2 is the file path and name of second set, minus the number
 *            Use either this or -f
 *            Set by command param -fd [int>0] [string] [string]
 *
 *
 * Input Assumptions:
 *    Each input file will represent 1 Fundamental Subset
 *    Each input file will have the same dimensions 
 *         (Vectors per subset, features per vector, total size)
 *    User will provide the number of elements per feature vector at run time
 *
 *******************************************************************************/
int main(int argc, const char ** argv) {

    bool gpuDevice = false; //Tracks whether a GPU device is available or not
    bool useCPU = false;    //Perform calculations on CPU?
    bool useGPU = true;     //Perform calculations on GPU? (default)

//--------------------------------------------------------------------------------------------------
//This section of code deals with input parameters from the command line

    //Initialize default option values
    unsigned device = 0; //ID of GPU to run on 
    bool discretize_input = true; //Discretize the input files
    bool discrete_output = false; //Discretize the output files
    unsigned num_bins = 15; //Discrete false by default, this is for simplifying testing
    bool time_code = false; //Time the code
    bool verbose_info = false; //Print calculation details
    bool measure_within_set = false; //Include or exclude single family intersections
	bool metricOnGPU = false;	//Run metrics on GPU rather than CPU.
	bool metricOnCPU = false;	//Run metrics on CPU for d-iterated pseudometric.
	metric_t<float> embeddedMetric = p_no_embeddedMetric<float>;	//Metric to be embedded into a pseudometric for CPU, if required (none by default)
	pseudometric_t<float> pseudometric;	//Pseudometric that the d-iterated pseudometric will utilize for CPU
	metric_t<float>* embeddedMetricGPU = &p_no_embeddedMetric<float>;		//Metric to be embedded into a pseudometric for CPU, if required (none by default)
	pseudometric_t<float>* pseudometricGPU;	//Pseudometric that the d-iterated pseudometric will utilize for CPU

    std::string file_pattern; //Name of files for the input data
    int setA_index; //Index of sets for set family A
	int setB_index; //Index of sets for set family B
    std::vector<std::string> fileName; //Store list of input files

    //Set option values for each parameter entered
    for(unsigned i = 0; i < argc; ++i){
        if(argv[i] == std::string("-c")){
            //Set program to perform on CPU only
            useCPU = true;
            useGPU = false;

        }else if(argv[i] == std::string("-cg")){
            //Set program to perform on GPU then CPU
            useCPU = true;

        }else if(argv[i] == std::string("-d")){
            //Instruct program to discretize input
            discretize_input = true;

        }else if(argv[i] == std::string("-b")){
            //Set how many bins to discretize into
            std::stringstream convert(argv[i + 1]);
            convert >> num_bins;
            i++;

        }else if(argv[i] == std::string("-mt")){
            //Indicate whether the GPU code should check for empty sets or not
            emptySetCheck = true;

        }else if(argv[i] == std::string("-gpu")){
            //Which device to use
            std::stringstream convert(argv[i + 1]);
            convert >> device;
            i++;

        }else if(argv[i] == std::string("-t")){
            time_code = true;

        }else if(argv[i] == std::string("-cores")){
            //How many cores the cpu has
            std::stringstream convert(argv[i + 1]);
                convert >> CORES;
                if(CORES < 1)
                    CORES = 1;
            i++;

        }else if(argv[i] == std::string("-o")){
            //Declare the size of the feature vectors
            std::stringstream convert(argv[i + 1]);
            convert >> VECTOR_SIZE;
            i++;

        }else if(argv[i] == std::string("-md")){
            //Declare the maximum depth of intersections
            std::stringstream convert(argv[i + 1]);
            convert >> maxDepth;
            i++;

        }else if(argv[i] == std::string("-v")){
            verbose_info = true;

        }else if(argv[i] == std::string("-do")){
            discrete_output = true;

        }else if(argv[i] == std::string("-fd")){
            std::stringstream convert(argv[i + 1]);
			convert >> SETS;
			std::stringstream convertA(argv[i + 3]);
			convertA >> setA_index;
			std::stringstream convertB(argv[i + 4]);
			convertB >> setB_index;
            file_pattern = argv[i+2];
            i+=4;

        }else if(argv[i] == std::string("-help")){
            printHelp();
            return(0);
        
        }else if(argv[i] == std::string("-in")){
            measure_within_set = true;
            
        }else if(argv[i] == std::string("-f")){
            //Push all files after -f into fileName vector
            for(unsigned j = i + 1; j<argc; ++j){
                fileName.push_back(argv[j]);
            }
            break;
		}else if (argv[i] == std::string("-dip")) {
			metricOnCPU = true;
			maxDepth = 1;
			if (argv[i + 1] == std::string("-djd")) {
				pseudometric = descJaccardDistance<float>;
				i++;
			}
			else if (argv[i + 1] == std::string("-dhd")) {
				pseudometric = descHausdorffDistance<float>;
				PSEUDOMETRIC_USES_DESCRIPTIVE_INTERSECTIONS = false;
				if (argv[i + 2] == std::string("-vhd")) {
					embeddedMetric = vectorHammingDistance<float>;
					i += 2;
				} else {
					std::cerr << "Please use a supported metric to embed into Hausdorff!" << endl;
					exit(1);
				}
			} else {
				std::cerr << "Please use a supported pseudometric!" << endl;
				exit(1);
			}
		}else if(argv[i] == std::string("-dipgpu")) {
			metricOnGPU = true;
			maxDepth = 1;
			if (argv[i + 1] == std::string("-djd")) {
				pseudometricGPU = &p_descJaccardDistance<float>;
				i++;
			}
			else if (argv[i + 1] == std::string("-dhd")) {
				std::cerr << "Due to a bug, Descriptive Hausdorff Distance is not functional when ran on the GPU!  Please use the CPU version." << endl;
				exit(1);
			}
			else {
				std::cerr << "Please use a supported pseudometric!" << endl;
				exit(1);
			}
        }else if(i > 0){
            std::cout << "Unknown parameter " << argv[i] << 
                ", use -help for a list of possible parameters.\n";
        }
    }

    //Check for valid VECTOR_SIZE
    if(VECTOR_SIZE < 1){
        std::cerr << "The number of elements in each feature vector must be > 0. " <<
            "Set this with the -o parameter.\n";
        exit(1);
    }

    if(fileName.size() == 0){
        string fileNumber;
		
		for(unsigned i = 0; i < SETS/2; i++){
			std::string fileString = file_pattern;
            ostringstream convert;
            convert << (setA_index + i);
            fileNumber = convert.str();
            fileName.push_back(fileString.append(fileNumber).append(".txt"));
        }
		
		for(unsigned i = 0; i < SETS/2; i++){
			std::string fileString = file_pattern;
            ostringstream convert;
            convert << (setB_index + i);
            fileNumber = convert.str();
            fileName.push_back(fileString.append(fileNumber).append(".txt"));
        }
    }

    //Number of Fundamental Subsets is equal to the number of input files
    F_SUBSET_COUNT = fileName.size();
	SUBSETS_PER_FAMILY = F_SUBSET_COUNT / 2;

    //Throw error if discretize option is chosen, but invalid, or no bin count is supplied
    if(discretize_input == true && num_bins < 1){
            std::cerr << "Use of -d requires -b  > 0\n";
            exit(1);
    }

    //If maxDepth wasn't set (or is invalid) set to F_SUBSET_COUNT (max depth)
    if(maxDepth <= 0 || maxDepth > F_SUBSET_COUNT)
        maxDepth = F_SUBSET_COUNT;

//--------------------------------------------------------------------------------------------------
    CUDA_CHECK_RETURN(hipSetDevice(device));

    size_t deviceMemory;
    //Get information about the available devices
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for(int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        if(verbose_info == true){
            printf("Device Number: %d\n", i);
            printf("Device name: %s\n", prop.name);
			printf("Shared Memory Limit: %d\n", prop.sharedMemPerBlock);
            printf("Potential tile width: %f!\n", min( 512.0,pow((float)2,
                      floor(log2f(prop.sharedMemPerBlock / ((VECTOR_SIZE + 1)*sizeof(float)*2))))));
        }
        //prop.major checks if the device is emulated or not. 
        //If not emulated, a gpu device is available
        if(prop.major != 9999) 
            gpuDevice = true;
        //Set TILE_WIDTH to the max the specified device can handle
        if(device == i){
            TILE_WIDTH = min(512.0,pow((float)2,floor(log2f(prop.sharedMemPerBlock / 
                             ((VECTOR_SIZE + 1) * sizeof(float) * 2)))));
            deviceMemory = prop.totalGlobalMem;
        }
    }
    std::cout << "Using device " << device << std::endl;
    
    //If no GPU is available, use the CPU
    if(gpuDevice==false) 
        useCPU = true;

    //Total number of elements across all Fundamental Subsets, does not include the Count attribute
    unsigned totalSize = 0;

    std::fstream myfile(fileName[0].c_str(), std::ios_base::in);

    if(myfile.fail()){
        cerr << "Error: " << fileName[0].c_str() << " could not be found.\n";
        exit(1);
    }

    float fileElement;
    //Get size of file
    while (myfile >> fileElement) {
        totalSize++;
    }
    //Get total size of all files combined
    totalSize *= F_SUBSET_COUNT;
    myfile.close();

    //Number of Feature Vectors in each Fundamental Subset
    VECTORS_PER_SUBSET = (totalSize / VECTOR_SIZE) / F_SUBSET_COUNT;

    if(verbose_info == true)
        printf("(Vectors per Set %i)  (Total Size %i) (Vector Size %i)  (Set Count %i)\n", 
                   VECTORS_PER_SUBSET, totalSize/F_SUBSET_COUNT, VECTOR_SIZE, F_SUBSET_COUNT);

    //Calculate how many sets can be handles, given the size of each, and desired intersection depth
    deviceMemory = deviceMemory / (VECTORS_PER_SUBSET * (VECTOR_SIZE+1) * sizeof(float));
    unsigned possibleSets = 1;
    while(true){
        unsigned x = 1;
        bitString pascalSum = 1;
        for(unsigned i = 0; i < maxDepth; ++i){
            if(i>possibleSets) break;
            x = x * ((possibleSets + 1 - i) / (i + 1.0));
            pascalSum += x;
        }
        if(pascalSum > deviceMemory)
            break;
        possibleSets++;
    }

    //Lower possible sets to account for other data usage
    possibleSets = possibleSets - (2 - (possibleSets % 2));
    //Max possible sets is 64 (# of bits in an unsigned long long int)
    possibleSets = min(64, possibleSets);

    printf("With depth %i, you can handle %i sets!\n",maxDepth, possibleSets);

    //If user is trying to run too many files, exit program
    if(F_SUBSET_COUNT > possibleSets){
        std::cerr << "Not enough memory for " << F_SUBSET_COUNT <<" sets of " << VECTOR_SIZE << 
            "x" << VECTORS_PER_SUBSET << " elements\n";
        exit(1);
    }

	if (metricOnCPU || metricOnGPU) {
		if (metricOnCPU && metricOnGPU) {
			std::cerr << "Metric must be run on CPU or GPU, but not both.";
			exit(1);
		}
		//read input into arrays
		float* family_A = new float[totalSize / 2];
		float* family_B = new float[totalSize / 2];
		float* discretizedInput = new float[totalSize];

		for (unsigned i = 0; i < F_SUBSET_COUNT; ++i) {
			std::fstream inputFile(fileName[i].c_str(), std::ios_base::in);
			if (inputFile.fail()) {
				cerr << "Error: File " << fileName[i].c_str() << " could not be found.\n";
				exit(1);
			}
			for (unsigned j = 0; inputFile >> fileElement; j++) {
				discretizedInput[j + (i * VECTOR_SIZE * VECTORS_PER_SUBSET)] = fileElement;
			}
		}
		if (discretize_input == true) {
			discretize(discretizedInput, totalSize, num_bins);
		}
		std::copy(discretizedInput, discretizedInput + (totalSize / 2), family_A);
		std::copy(discretizedInput + (totalSize / 2), discretizedInput + totalSize, family_B);
		delete[] discretizedInput;

		float result;
		if (metricOnCPU) {
			result = dIteratedPseudometric<float>(family_A, family_B, time_code, pseudometric, embeddedMetric);
		} else {
			result = dIteratedPseudometricGPU<float>(family_A, family_B, time_code, pseudometricGPU, embeddedMetricGPU);
		}

		printf("\nDescriptive Set Intersections final Measure: %f\n", result);

		delete[] family_A;
		delete[] family_B;
		return 0;
	}


    //Get pascalMax, the highest pascal number of the F_SUBSET_COUNT-th row,
    // pascalMax also stores how many parallel streams are needed
    //Get pascalTotal, a weighted sum of pascal numbers, used to calculate final measure
    //Initialize totalMeasure, which holds the sum of all weighted measures
    float *prefixPascal = new float[maxDepth+1];
    prefixPascal[0] = 0;
    prefixPascal[1] = 1;
    bitString pascalMax = 0;
    float totalMeasure = 0;
    bitString pascalTotal = F_SUBSET_COUNT;
    bitString emptySetSize = 1;

    float x = F_SUBSET_COUNT;
    for(unsigned i = 1; i <= maxDepth; ++i){
        emptySetSize += x;
        if(i < maxDepth){
            prefixPascal[i+1] = x + prefixPascal[i];
        }
        if(x > pascalMax)
            pascalMax = x;
        
        x = x * ((F_SUBSET_COUNT - i)/ (i + 1.0));
        if(i<maxDepth){
            pascalTotal += (1 + i) * (1 + i) * x;
        }
    }

    //Total width of the intersections power set array
    WIDTH = emptySetSize * VECTORS_PER_SUBSET;

    //Declare array to be discretized, of size, filesize
    float *fundamentalSubset = new float[totalSize];
    float *originalValues;
    if(discrete_output == false)
        originalValues = new float[totalSize];

    //Fill in array with values from input file
    for(unsigned i = 0; i < F_SUBSET_COUNT; ++i){
        unsigned z = 0;
        std::fstream inputFile(fileName[i].c_str(), std::ios_base::in);
        if(inputFile.fail()){
            cerr << "Error: File " << fileName[i].c_str() << " could not be found.\n";
            exit(1);
        }

        while (inputFile >> fileElement) {
            fundamentalSubset[(i * VECTORS_PER_SUBSET) + (z / VECTOR_SIZE) + ((z % VECTOR_SIZE) * 
                VECTORS_PER_SUBSET * F_SUBSET_COUNT)] = fileElement;
            //If user wants original values output, store them in a seperate array
            if(discrete_output == false)
                originalValues[(i * VECTORS_PER_SUBSET) + (z / VECTOR_SIZE) + ((z % VECTOR_SIZE) * 
                    VECTORS_PER_SUBSET * F_SUBSET_COUNT)] = fileElement;
            z++;
        }
        z = 0;
    }

    //Call function to discretize, if specified by user
    if(discretize_input == true)
        discretize(fundamentalSubset, totalSize, num_bins);
        
    //Declare array to hold the Set Descriptions of each Fundamental Subset
    bitString interSetSize = WIDTH * (VECTOR_SIZE + 1);
    float *intersections = new float[interSetSize];
    CUDA_CHECK_RETURN(hipMallocManaged(&intersections, interSetSize * sizeof(float)));
    printf("Malloc Intersection Set: %s \n", hipGetErrorString(hipGetLastError()));

    //Jump here after GPU calculation, if performing calculation on both CPU and GPU (for testing)
    rerunOnCPU:

    //Set all values in the Set Description array to the minimum float value
    initNegative(intersections, WIDTH * (VECTOR_SIZE + 1));

    //Create the Set Descriptions and save them into the Set Description array
    createSetDescription(fundamentalSubset, intersections);

    //Run the intersections on the GPUs
    //If no GPUs are found, or the user specifies to not use them, run the intersections on the CPU
    if(useGPU == true){
        gpuIntersections(intersections, prefixPascal, time_code, emptySetSize);
    }

    if(useCPU == true){
        if(useGPU == true){
            //If user instructed to use CPU and GPU, clear results and run CPU code
            useGPU = false;
            goto rerunOnCPU;
        }
        cpuIntersections(intersections, prefixPascal, time_code);
    }

    //Calculates the final measure of the closeness of intersections
    totalMeasure = calculateMeasure(emptySetSize, prefixPascal, intersections, pascalTotal, 
                       verbose_info, measure_within_set);

    //Write output of final intersection to file
    if(discrete_output == true){
        writeToFile_D(intersections, emptySetSize);
    }else{
        writeToFile(intersections, originalValues);
    }
    
    //Print final measure of nearness of sets
    printf("\nDescriptive Set Intersections final Measure: %f\n", totalMeasure);
	
    hipFree(intersections);
    return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line, 
                                  const char *statement, hipError_t err){
    if (err == hipSuccess)
        return;
    std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at " 
        << file << ":" << line << std::endl;
    exit (1);
}
