#include "hip/hip_runtime.h"
/*
====================================================================================================
 Name        : DeTopS.cu
 Author      : Jesse Harder
 Supervisor  : Dr. Christopher Henry, P. Eng.
 Date        : Sept 16, 2018
 Version     : 2.1
 Modified    : Jesse Harder
 Description : This program will:
                    -Optionally discretize input data, from multiple files
                    -Develop set descriptions for each input set
                    -Perform the descriptive intersection power set for the set of input files (CPU or GPU)
                    -Calculate a measure of the closeness of the sets intersected
                    -Output the results of the intersections to a text file
License     : Licensed under the Non-Profit Open Software License version 3.0
 1) Grant of Copyright License. Licensor grants You a worldwide, royalty-free,
 non-exclusive, sublicensable license, for the duration of the copyright, to do the following:

 a) to reproduce the Original Work in copies, either alone or as part of a collective work;

 b) to translate, adapt, alter, transform, modify, or arrange the Original Work, thereby
 creating derivative works ("Derivative Works") based upon the Original Work;

 c) to distribute or communicate copies of the Original Work and Derivative Works
 to the public, with the proviso that copies of Original Work or Derivative Works
 that You distribute or communicate shall be licensed under this Non-Profit Open Software
 License or as provided in section 17(d);

 d) to perform the Original Work publicly; and

 e) to display the Original Work publicly.

 2) Grant of Patent License. Licensor grants You a worldwide, royalty-free,
 non-exclusive, sublicensable license, under patent claims owned or controlled by
  the Licensor that are embodied in the Original Work as furnished by the Licensor,
  for the duration of the patents, to make, use, sell, offer for sale, have made,
   and import the Original Work and Derivative Works.

 3) Grant of Source Code License. The term "Source Code" means the preferred
 form of the Original Work for making modifications to it and all available
 documentation describing how to modify the Original Work. Licensor agrees to
 provide a machine-readable copy of the Source Code of the Original Work along
 with each copy of the Original Work that Licensor distributes. Licensor reserves
 the right to satisfy this obligation by placing a machine-readable copy of the
 Source Code in an information repository reasonably calculated to permit
 inexpensive and convenient access by You for as long as Licensor continues
 to distribute the Original Work.

 4) Exclusions From License Grant. Neither the names of Licensor, nor the names
 of any contributors to the Original Work, nor any of their trademarks or service
 marks, may be used to endorse or promote products derived from this Original Work
 without express prior permission of the Licensor. Except as expressly stated
  herein, nothing in this License grants any license to Licensor's trademarks,
  copyrights, patents, trade secrets or any other intellectual property. No patent
  license is granted to make, use, sell, offer for sale, have made, or import embodiments
  of any patent claims other than the licensed claims defined in Section 2. No license
 is granted to the trademarks of Licensor even if such marks are included in the Original
  Work. Nothing in this License shall be interpreted to prohibit Licensor from licensing
  under terms different from this License any Original Work that Licensor otherwise would
  have a right to license.

 5) External Deployment. The term "External Deployment" means the use, distribution, or
 communication of the Original Work or Derivative Works in any way such that the Original
 Work or Derivative Works may be used by anyone other than You, whether those works are
 distributed or communicated to those persons or made available as an application intended
 for use over a network. As an express condition for the grants of license hereunder,
 You must treat any External Deployment by You of the Original Work or a Derivative
 Work as a distribution under section 1(c).

 6) Attribution Rights. You must retain, in the Source Code of any Derivative Works
 that You create, all copyright, patent, or trademark notices from the Source Code of
 the Original Work, as well as any notices of licensing and any descriptive text
 identified therein as an "Attribution Notice." You must cause the Source Code for
 any Derivative Works that You create to carry a prominent Attribution Notice reasonably
 calculated to inform recipients that You have modified the Original Work.

 7) Warranty of Provenance and Disclaimer of Warranty. The Original Work is provided
 under this License on an "AS IS" BASIS and WITHOUT WARRANTY, either express or implied,
 including, without limitation, the warranties of non-infringement, merchantability or
 fitness for a particular purpose. THE ENTIRE RISK AS TO THE QUALITY OF THE ORIGINAL WORK
 IS WITH YOU. This DISCLAIMER OF WARRANTY constitutes an essential part of this License.
 No license to the Original Work is granted by this License except under this disclaimer.

 8) Limitation of Liability. Under no circumstances and under no legal theory, whether
 in tort (including negligence), contract, or otherwise, shall the Licensor be liable
 to anyone for any direct, indirect, special, incidental, or consequential damages of
 any character arising as a result of this License or the use of the Original Work
 including, without limitation, damages for loss of goodwill, work stoppage, computer
 failure or malfunction, or any and all other commercial damages or losses. This limitation
 of liability shall not apply to the extent applicable law prohibits such limitation.

 9) Acceptance and Termination. If, at any time, You expressly assented to this License,
 that assent indicates your clear and irrevocable acceptance of this License and all of
 its terms and conditions. If You distribute or communicate copies of the Original Work
 or a Derivative Work, You must make a reasonable effort under the circumstances to obtain
 the express assent of recipients to the terms of this License. This License conditions
 your rights to undertake the activities listed in Section 1, including your right to create
 Derivative Works based upon the Original Work, and doing so without honoring these terms and
 conditions is prohibited by copyright law and international treaty. Nothing in this License
 is intended to affect copyright exceptions and limitations (including "fair use" or "fair
 dealing"). This License shall terminate immediately and You may no longer exercise any of
 the rights granted to You by this License upon your failure to honor the conditions in Section 1(c).

 10) Termination for Patent Action. This License shall terminate automatically and You
 may no longer exercise any of the rights granted to You by this License as of the date
 You commence an action, including a cross-claim or counterclaim, against Licensor or any
 licensee alleging that the Original Work infringes a patent. This termination provision
 shall not apply for an action alleging patent infringement by combinations of the Original
  Work with other software or hardware.

 11) Jurisdiction, Venue and Governing Law. Any action or suit relating to this License
 may be brought only in the courts of a jurisdiction wherein the Licensor resides or in
 which Licensor conducts its primary business, and under the laws of that jurisdiction
 excluding its conflict-of-law provisions. The application of the United Nations Convention
 on Contracts for the International Sale of Goods is expressly excluded. Any use of the Original
 Work outside the scope of this License or after its termination shall be subject to the
 requirements and penalties of copyright or patent law in the appropriate jurisdiction.
 This section shall survive the termination of this License.

 12) Attorneys' Fees. In any action to enforce the terms of this License or seeking
 damages relating thereto, the prevailing party shall be entitled to recover its costs and
 expenses, including, without limitation, reasonable attorneys' fees and costs incurred in
 connection with such action, including any appeal of such action. This section shall survive
 the termination of this License.

 13) Miscellaneous. If any provision of this License is held to be unenforceable, such provision
 shall be reformed only to the extent necessary to make it enforceable.

 14) Definition of "You" in This License. "You" throughout this License, whether in upper or
 lower case, means an individual or a legal entity exercising rights under, and complying with
 all of the terms of, this License. For legal entities, "You" includes any entity that controls,
 is controlled by, or is under common control with you. For purposes of this definition, "control"
 means (i) the power, direct or indirect, to cause the direction or management of such entity,
 whether by contract or otherwise, or (ii) ownership of fifty percent (50%) or more of the outstanding
 shares, or (iii) beneficial ownership of such entity.

 15) Right to Use. You may use the Original Work in all ways not otherwise restricted or conditioned
  by this License or by law, and Licensor promises not to interfere with or be responsible for such uses by You.

 16) Modification of This License. This License is Copyright © 2005 Lawrence Rosen.
 Permission is granted to copy, distribute, or communicate this License without modification.
 Nothing in this License permits You to modify this License as applied to the Original Work or to
 Derivative Works. However, You may modify the text of this License and copy, distribute or communicate
 your modified version (the "Modified License") and apply it to other original works of authorship
 subject to the following conditions: (i) You may not indicate in any way that your Modified License
 is the "Open Software License" or "OSL" and you may not use those names in the name of your Modified
 License; (ii) You must replace the notice specified in the first paragraph above with the notice
 "Licensed under <insert your license name here>" or with a notice of your own that is not confusingly
 similar to the notice in this License; and (iii) You may not claim that your original works are open
 source software unless your Modified License has been approved by Open Source Initiative (OSI) and
 You comply with its license review and certification process.

 17) Non-Profit Amendment. The name of this amended version of the Open Software License ("OSL 3.0")
 is "Non-Profit Open Software License 3.0". The original OSL 3.0 license has been amended as follows:

 (a) Licensor represents and declares that it is a not-for-profit organization that derives no revenue
 whatsoever from the distribution of the Original Work or Derivative Works thereof, or from support
 or services relating thereto.

 (b) The first sentence of Section 7 ["Warranty of Provenance"] of OSL 3.0 has been stricken. For
 Original Works licensed under this Non-Profit OSL 3.0, LICENSOR OFFERS NO WARRANTIES WHATSOEVER.

 (c) In the first sentence of Section 8 ["Limitation of Liability"] of this Non-Profit OSL 3.0,
 the list of damages for which LIABILITY IS LIMITED now includes "direct" damages.

 (d) The proviso in Section 1(c) of this License now refers to this "Non-Profit Open Software
 License" rather than the "Open Software License". You may distribute or communicate the Original
 Work or Derivative Works thereof under this Non-Profit OSL 3.0 license only if You make the
 representation and declaration in paragraph (a) of this Section 17. Otherwise, You shall distribute or
 communicate the Original Work or Derivative Works thereof only under the OSL 3.0 license and You shall
 publish clear licensing notices so stating. Also by way of clarification, this License does not authorize
 You to distribute or communicate works under this Non-Profit OSL 3.0 if You received them under
 the original OSL 3.0 license.

 (e) Original Works licensed under this license shall reference "Non-Profit OSL 3.0"
 in licensing notices to distinguish them from works licensed under the original OSL 3.0 license.
====================================================================================================
*/
//---------------------------------------------------------------------------------
#include <iostream>   //Standard input output
#include <fstream>    //Read input and write output files
#include <vector>     //Provides access to vector object, for flexibly sized arrays
#include <math.h>     //Provides math functions. pow, log, ceil, floor
#include <stdlib.h>   //Provides size_t datatype
#include <string>     //Provides string object
#include <sstream>    //Provides methods for working with strings
#include <limits>     //Used to derive minFloat
#include <ctime>      //Used for CPU timing code
#include <pthread.h>  //Used for parallel CPU threads
#include <mutex>      //Used for synchronization of parallel cpu code
//---------------------------------------------------------------------------------
static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

unsigned SETS = 10;    //How many subsets to load in (for testing)
#define STREAMS 500    //How many streams to launch intersectKernels in
typedef unsigned long long bitString;

bool emptySetCheck = false;
//Most negative float value, used as a null in arrays
const float minFloat = (-1) * (std::numeric_limits<float>::max());
//Maximum depth of intersections (max number of sets that can take place in an intersection)
unsigned maxDepth = 0;       
unsigned F_SUBSET_COUNT = 0;  //Number of input sets
unsigned VECTORS_PER_SUBSET;  //Width of each fundamental subset
unsigned VECTOR_SIZE;         //Features per feature vector, defines shared memory tile length
unsigned WIDTH;               //Total width of the output set
unsigned CORES = 1;           //How many cores to run cpu on
unsigned TILE_WIDTH;          //Tile width of intersectKernel

//Global variables used for parallel CPU intersection code
bitString bitPermute;
bitString bitCount;
unsigned cpuDepth = 0;
std::mutex mtx;

using namespace std;

/**
 * This structure is used for passing multiple arguments to the 
 * CPU Intersection function (intersectCPU)
 *     prefixes: A prefix summed set of a row of Pascal's Triangle
 *       pascal: The current pascal number (how many intersections to perform)
 *            a: A pointer to the intersections array (input and output)
 */
typedef struct{
    unsigned pascal;
    float *a;
    float *prefixes;
} intersectArgs;

/******************************************************************************
 * isEmptyKernel
 *
 * This function will determine if a set in the intersections set is the empty set
 * If a thread encounters a non (minFloat) value, it writes a 1 to that intersections
 * position in b, indicating that some non-empty results are in the intersection
 * [in]:
 *         a: The set containing the results of all performed intersections
 *         b: A set containing one value for each set/intersection in a
 *         index: The location in a of the subset to be checked
 *         VECTORS_PER_SUBSET: How many values need to be checked in b
 *         minFloat: The most negative float value, signifies a null or empty result
 *
 * [out]:
 *        b: Modified to have a 1 in the position of every non-empty set
 * [return]:
 *        isEmpty: True if the first value of every vector in the subset is minFloat
 *
 *****************************************************************************/
__global__ void isEmptyKernel(float* a, float *b, bitString index, unsigned VECTORS_PER_SUBSET, 
                                  float  minFloat) {

    //Tracks if any thread in block has found a non empty vector
    __shared__ bool isNotEmpty;

    unsigned id = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (a[VECTORS_PER_SUBSET * index + id] > minFloat && id < VECTORS_PER_SUBSET) {
        isNotEmpty = true;
    }
    __syncthreads();

    //if non-empty vector is found, thread 0 writes to the output array
    if(isNotEmpty == true && threadIdx.x == 0){
        b[index] = 1;
    }
}

/******************************************************************************
 * isEmptySet
 *
 * This function will determine if a set in the intersections set is the empty set
 *
 * [in]:
 *         a: The set containing the results of all performed intersections
 *         index: The location in a of the subset to be checked
 *
 * [return]:
 *         isEmpty: True if the first value of every vector in the subset is minFloat
 *
 *****************************************************************************/
 bool isEmptySet(float *a, bitString index){

    for(unsigned i=0; i < VECTORS_PER_SUBSET; i++){
        if(a[index + i] > minFloat){
            return false;
        }
    }
    return true;
}

/******************************************************************************
 *
 * intersectKernel
 *
 * Each thread intersects an object from one set with all of the objects in another set
 * Objects found in both sets are printed out
 *
 * [in]:
 *         *a: An array containing all sets and vectors
 *         F_SUBSET_SIZE: Number of fundamental subsets in the input
 *         VECTORS_PER_SUBSET: The number of objects per set
 *         VECTOR_SIZE: Number of elements in each vector
 *         indexA: The array index for set A
 *         indexB: The array index for set B
 *         minFloat: Lowest float value, used for "null" data
 *
 * [out]:
 *         A set of all feature vectors that appear in both a[index] 
 *         and a given vector in intersectionSet
 *
 * [return]:
 *         Void
 *
 *******************************************************************************/
__global__ void intersectKernel(float *a, unsigned F_SUBSET_COUNT, unsigned VECTORS_PER_SUBSET, 
                                    unsigned VECTOR_SIZE, bitString indexA, bitString indexB, 
                                    bitString indexC, float minFloat, unsigned WIDTH){

    //Shared memory to store the two shared memory matrices, A B
    extern __shared__ float tiles[];
    unsigned TILE_WIDTH = blockDim.x;
    float *tileA = &tiles[0];
    float *tileB = &tiles[(TILE_WIDTH * (VECTOR_SIZE + 1))];

    //Boolean  that tracks if this thread's vector has matched with any vector in the other set
    bool inIntersect = false;
    unsigned tx = threadIdx.x;
    unsigned width = WIDTH;

    //Overall id of current thread
    unsigned id = (blockDim.x * blockIdx.x) + threadIdx.x;

    //Load this thread's vector of a into shared memory
    for(unsigned i = 0; i < VECTOR_SIZE + 1; ++i){
        if(id < VECTORS_PER_SUBSET){
            tileA[i*TILE_WIDTH + tx] = a[indexA * VECTORS_PER_SUBSET + id + width * i];
        }else{
            //Set element to minimum value (value to be ignored), if the thread is out of bounds
            tileA[i*TILE_WIDTH + tx] = minFloat;
        }
    }

    for(unsigned q = 0; q < gridDim.x; ++q){
        //Load this thread's corresponding vector in the qth tile of b into shared memory
        for(unsigned i = 0; i < VECTOR_SIZE + 1; ++i){
            if((q * TILE_WIDTH) + tx < VECTORS_PER_SUBSET){
                tileB[i*TILE_WIDTH + tx] = 
                    a[indexB * VECTORS_PER_SUBSET + tx + (TILE_WIDTH * q) + (width * i)];
            }else{
                //Set element to minimum value (value to be ignored), if the thread is out of bounds
                tileB[i*TILE_WIDTH + tx] = minFloat;
            }
        }

        __syncthreads();

        if(tileA[tx] > minFloat && inIntersect == false){

            for(unsigned i=0; i < TILE_WIDTH; ++i){
                //Two vectors are equal until non-equal elements in the vectors are encountered
                bool match = true;

                for(unsigned j=0; j < VECTOR_SIZE; ++j){
                ////! Replace this if statement with a function to suit your implementation!////
                    if(tileA[j*TILE_WIDTH + tx] != tileB[j*TILE_WIDTH + i]){
                        match = false;
                        break;
                    }
                }

                if(match == true){
                    inIntersect = true;
                    tileA[(VECTOR_SIZE)*TILE_WIDTH + tx] += tileB[(VECTOR_SIZE)*TILE_WIDTH + i];
                    break;
                }
            }

        }
        __syncthreads();
    }

    //If this vector has not matched with any vector in the other set, set it to "null"
    if(inIntersect == false){
        for(unsigned i = 0; i < VECTOR_SIZE + 1; ++i){
            tileA[i*TILE_WIDTH + tx] = minFloat;
        }
    }

    __syncthreads();

    //Write tileA to it's space in a (the intersection power set)
    if(id < VECTORS_PER_SUBSET){
        for(unsigned i = 0; i < VECTOR_SIZE + 1; ++i){
            a[indexC * VECTORS_PER_SUBSET + id + width * i] = tileA[i*TILE_WIDTH + tx];
        }
    }
}

/******************************************************************************
 * getTrailingZeros
 *
 * Calculates how many trailing 0s exists in the binary form of a number
 *         This function is to be called as a part of function: next_perm
 * [in]:
 *         w: Some integer to be checked
 * [out]:
 *         None
 * [return]:
 *         x: The count of the trailing zeros of w
 *
 *******************************************************************************/
bitString getTrailingZeros(bitString w){
    bitString x = 0;
    while(w % 2 == 0 && w > 0){
        w = w >> 1;
        x++;
    }
    return x;
}

/******************************************************************************
 * next_perm
 * Gives the next permutation for a bit sequence containing the same number of bits as v
 * [in]: v: previous permutation
 * [out]: none
 * [return]: Next permutation for a bit sequence containing the same number of bits as v
 * [comments]: Used in conjunction with element_0. Function obtained from:
 *               http://alexbowe.com/popcount-permutations/
 *               http://graphics.stanford.edu/~seander/bithacks.html
 *
 *******************************************************************************/
bitString next_perm(bitString v){
    //v is the current permutation of bits
    bitString w; //next permutation of bits

    bitString t = v | (v - 1); // t gets v's least significant 0 bits set to 1
    //Next set to 1 the most significant bit to change,
    //set to 0 the least significant ones, and add the necessary 1 bits.
    w = (t + 1) | (((~t & -~t) - 1) >> (getTrailingZeros(v) + 1));

    return w;
}

/******************************************************************************
 * getBitPatternIndex
 *
 * This function returns a number corresponding to which bit pattern the input is at a given level
 * For example, 0011 is the first pattern of 2 bits, 0101 the second , 0110 the third...
 *
 *
 * [in]:
 *         depth: the number of on bits in the bit pattern
 *         bitPattern: a string of bits with depth 1's
 *
 * [out]:
 *        None
 *
 * [return]:
 *         i: the index of which combination of #depth bits was provided
 *
 *******************************************************************************/
bitString getBitPatternIndex(unsigned depth, bitString bitPattern){
    unsigned i = 0; //Stores the current index of the bit pattern 
    bitString compareBits = (1 << depth) -1; //Get first bit pattern with #depth one's
    while( true ){
        if(bitPattern == compareBits)
            return i;
        i++;
        compareBits = next_perm(compareBits);
    }
}

/******************************************************************************
 * intersectCPU
 *
 * This function will intersect two sets together using the CPU.
 * Each thread of this function will handle an intersection.
 * When the intersection is completed, the thread will check if there are more intersections to do
 * if not, exit function
 *
 * [in]:
 *         args: An object containing:
 *            a: The intersections set, where data is read from, and written to
 *       pascal: The Pascal number stating how many intersections need to be performed at this level
 *     prefixes: A prefix sum of the pascal numbers, used to find indexes
 * [out]:
 *        This function will insert the result of the intersection to the intersection array
 *        at set index singleBit+myBits
 *
 * [return]:
 *         Void
 *******************************************************************************/
void *intersectCPU(void* args){
    bitString curBits;     //Bit index of the result of the intersection
    bitString myIndex;     //Index of Set A in intersections array
    bitString singleIndex; //Index of Set B in intersections array
    bitString outIndex;    //Index of output Set C in intersections array

    intersectArgs *arg = (intersectArgs*)args; //Holds data structure defined above
    std::unique_lock<std::mutex> critZone (mtx, std::defer_lock); //Declare lock for parallelization

    //Loop runs until there are no more intersections to be performed at this level
    while(true){
    ///////////////only one thread may perform this section at a time////////////////////
        critZone.lock();

        //If all intersections have been performed, exit function
        if(bitCount >= arg->pascal){
            free(arg);
            critZone.unlock();
            return 0;
        }

        //Get the next intersection to be performed
        if(bitCount > 0){
            bitPermute = next_perm(bitPermute);
        }
        curBits = bitPermute;
        outIndex = arg->prefixes[ 1 + cpuDepth ] + bitCount;
        singleIndex = 1+getBitPatternIndex(1, curBits & -curBits);
        myIndex = arg->prefixes[cpuDepth] + 
                      getBitPatternIndex(cpuDepth, curBits - (curBits & -curBits));
        bitCount++;
        critZone.unlock();
    ///////////////end single thread critical section //////////////////////////////////////

        //Determine which sets are being intersected
        outIndex *= VECTORS_PER_SUBSET;
        myIndex *= VECTORS_PER_SUBSET;
        singleIndex *= VECTORS_PER_SUBSET;

        //Tracks how many equal vectors have been found, 
        //This is used as an index for writing matched vectors to the Power Set array
        unsigned count=0;

        //If the set to be intersected is the empty set, skip to next intersection
        if(emptySetCheck == true){
            if(isEmptySet(arg->a, myIndex) == true){
                continue;
            }
        }

        //for each vector in set a
        for(unsigned k = 0; k < VECTORS_PER_SUBSET; ++k){
            //Compare to each vector in set b
            for(unsigned i = 0; i < VECTORS_PER_SUBSET; ++i){
                //Match is true until a value that does not match between the two vectors is found
                bool match = true;
                //Compare all of the elements in the two vectors
                for(unsigned j = 0; j< VECTOR_SIZE; ++j){
                    //If two elements don't match, or the value is minFloat the match is false
                    if(arg->a[singleIndex + k + (WIDTH * j)] != arg->a[myIndex + i + (WIDTH *j)] 
                           || arg->a[singleIndex + k + (WIDTH * j)] <= minFloat){
                        match = false;
                        break;
                    }
                }
                //If no non-matching elements are found, 
                // write the matching vector to the Intersection Power Set array
                if(match == true){
                    for(unsigned m = 0; m < VECTOR_SIZE + 1; ++m){
                        arg->a[outIndex  + k + (WIDTH * m)] =
                            arg->a[singleIndex + k + (WIDTH*m)];

                        //Add the feature counts of the two matching vectors together
                        if(m == VECTOR_SIZE)
                            arg->a[outIndex + k + (WIDTH * m)] =
                                arg->a[singleIndex + k + (WIDTH * m)] +
                                arg->a[myIndex + i + (WIDTH * m)];
                    }
                    count++;
                    break;
                }
            }
        }
    }
}

/******************************************************************************
 * discretize
 *
 * Discretizes an array of floats (Values 1 to (specified number of values) )
 *
 * [in]:
 *         *data: pointer to the array of floats to be discretized
 *         size: the size of the data array
 *         num_bins: the number of discrete values to divide the data into,
 *                   determined by user input
 *
 * [out]:
 *         *data: Discretized array
 *
 * [return]:
 *         void
 *
 *******************************************************************************/
void discretize(float *data, unsigned size, unsigned num_bins) {

    float min = data[0];    //Records the maximum value of the input data
    float max = data[0];    //Records the minimum value of the input data
    float *bounds = new float[num_bins + 1];    //Calculates the boundary values of the array

    //Get max and min of data set
    for (unsigned i = 0; i < size; ++i) {
        if (data[i] < min)
            min = data[i];
        if (data[i] > max) {
            max = data[i];
        }
    }
    
    //If min < 0, then shift values to the right, so all are positive
    if (min < 0) {
        for (unsigned i = 0; i < size; ++i) {
            data[i] += min * (-1);
        }
    }

    //If max != 1, divide all values in range 0-1
    if (max != 1) {
        for (unsigned i = 0; i < size; ++i) {
            data[i] /= max;
        }
    }

    //Generate bounds for partition, based on int parts
    float partSize = 1.0 / num_bins;
    for (unsigned i = 0; i < num_bins + 1; ++i) {
        bounds[i] = i * partSize;
    }

    //Set each data value into it's value range
    for (unsigned i = 0; i < size; ++i) {
        if (data[i] == bounds[num_bins]) {
            data[i] = num_bins;
            continue;
        }
        for (unsigned j = 0; j < num_bins; ++j) {
            if (data[i] >= bounds[j] && data[i] < bounds[j + 1]) {
                data[i] = j + 1;
            }
        }
    }
}

/******************************************************************************
 * initNegative
 *
 * Initializes the values in a float array to the lowest float value
 * These values serve as a check for `null` values in the set descriptions
 * The values in the last row are set to 1`s, the initial value of the object's 
 * frequency in the fundamental subset
 *
 * [in]:
 *         *data: The set description array, to be initialized
 *         size: The number of elements in the array, not including vector counts
 * [out]:
 *         *data: All values initialized to the most nagative float value
 *
 * [return]:
 *         Void
 *
*******************************************************************************/
void initNegative(float *data, unsigned size) {
    
    for (unsigned i = 0; i < size; ++i) {
        data[i] = minFloat;
    }
}

/******************************************************************************
 * createSetDescription
 *
 * Takes in a set of feature vectors, then finds all unique vectors in the set
 *
 *     [in]:
 *         *v: pointer to a set of feature vectors
 *         *w: pointer to output for set of unique descriptions
 *
 *     [out]:
 *         *w: vector of feature vectors to be filled with unique descriptions
 *
 *     [return]:
 *         void
 *
 *******************************************************************************/
void createSetDescription(float *v, float *w){

    for(int fa = 0; fa < F_SUBSET_COUNT; ++fa){
        unsigned setIndex = (fa + 1) * VECTORS_PER_SUBSET;
        int uniqueCount = 0;//Running total of the number of unique objects encountered
        
        //For each vector in A
        for(unsigned i = 0; i < VECTORS_PER_SUBSET; ++i){
            //Tracks if the current vector of v is unique (has not matched with any vectors of w)
            bool isUnique = true;
            //For each vector in D(A)
            for(unsigned j = 0; j < uniqueCount; ++j){
                //Tracks if the current vector of v matches with the current vector of w
                bool unique = false;
                //For each element in current vector
                for(unsigned k = 0; k < VECTOR_SIZE; k++){
                    //If any two elements don't match, then the two vectors don't match
                    if(v[(fa * VECTORS_PER_SUBSET) + (k * VECTORS_PER_SUBSET * F_SUBSET_COUNT) +i]
                          != w[setIndex + (k * WIDTH ) + j]){
                        unique = true;
                        break;
                    }
                }

                if(unique == false){
                    isUnique = false;
                    //If vector is not unique, increment the conut of the vector it matched with
                    w[setIndex + (VECTOR_SIZE * WIDTH ) + j]++;
                    break;
                }
            }
    
            if(isUnique){
                //If the vector is unique, insert it into intersection set
                for(int j = 0; j < VECTOR_SIZE; ++j){
                    w[setIndex + (j * WIDTH) + uniqueCount] = 
                       v[(fa * VECTORS_PER_SUBSET) + (j * VECTORS_PER_SUBSET * F_SUBSET_COUNT) +i];
                }
                w[setIndex + (VECTOR_SIZE * WIDTH) + uniqueCount] = 1;
                uniqueCount++;
            }
        }
    }
}

/******************************************************************************
 * calculateMeasure
 *
 * Calculates the final measure of closeness of sets
 *
 *     [in]:
 *         emptySetSize: the number of sets to be calculated
 *         *prefixPascal: an array of prefix summed Pascal numbers
 *         *intersections: the array of data to be operated upon
 *         pascalTotal: a weighted sum of pascal numbers, used for calculation
 *         verbose_info: a boolean specifying whether to print detailed info or not
 *         measure_within_set: a boolean specifying whether intersections of sets that are all 
 *                             within one family should be included or not
 *
 *     [out]:
 *         none
 *
 *     [return]:
 *         totalMeasure: the final result of the measure calculation
 *
 *******************************************************************************/
float calculateMeasure(unsigned emptySetSize, float* prefixPascal, float* intersections, 
                          float pascalTotal, bool verbose_info, bool measure_within_set){
    float totalMeasure = 0; //Stores total measure
    float weightedOut = 0;  //Total measure lost to intersections we don't want to include
    //how many sets are in each intersection
    unsigned depth = 0;
    unsigned checkPoint = 0;
	
    if(measure_within_set == true){
        printf("Include single family intersections in measure\n");
    }else{
        checkPoint = pow(2, F_SUBSET_COUNT/2);
        printf("Exclude single family intersections from measure: CheckPoint = %i\n", checkPoint);
    }

    for(bitString i = 1; i < emptySetSize; ++i){
        //Total count of vectors in this intersection
        float total = 0;
        bitString bitPattern;
        if(i == prefixPascal[depth+1]){
            depth++;
            bitPattern = (1 << depth) -1;
        }else{
            bitPattern = next_perm(bitPattern);
        }

        //Total all of the counts of vectors in this set
        for(unsigned j = 0; j < VECTORS_PER_SUBSET; ++j){
            if(intersections[(VECTORS_PER_SUBSET * i) + j + (WIDTH * VECTOR_SIZE)] > minFloat){
                total += intersections[(VECTORS_PER_SUBSET * i) + j + WIDTH * VECTOR_SIZE];
            }
        }
        //Calculate the weighted value of this set's count, and add it to the final measure
        float weightedValue = ((float)depth / pascalTotal) * (total / VECTORS_PER_SUBSET);
      
        //Ayotu
        if(measure_within_set == false && (bitPattern<checkPoint || (bitPattern%checkPoint == 0))){
            weightedOut += weightedValue;
            if(verbose_info == true) 
                printf("(Excluded from measure)");
        }else{
            totalMeasure += weightedValue;
        }
        //Print detailed information on each intersection performed
        if(verbose_info == true){
            std::cout << "Bit Pattern :" << bitPattern << ", ";
            printf("Index: %i, #Sets: %i  Count: %f / %i, Weighted: %f\n",
                       i, depth, total, depth * VECTORS_PER_SUBSET, weightedValue);
        }
    }

    //Remove the weight of single family intersections
    totalMeasure /= (1 - weightedOut);
	
    //Account for rounding
    if(totalMeasure > 1) totalMeasure = 1;

    //totalMeasure = (totalMeasure - ((float)F_SUBSET_COUNT / pascalTotal)) / 
    //                   (1 - ((float)F_SUBSET_COUNT / pascalTotal) );
    return totalMeasure;
}

/******************************************************************************
 * writeToFile
 *
 * Writes the results of the intersections to a text file
 *
 * [in]:
 *         *originalValue: an array of the original values read in from the input files
 *         *intersections: the array of data to be operated upon
 *
 *     [out]:
 *         result.txt: A text file containing all of the results from the intersections
 *
 *     [return]:
 *         void
 *
 *     [notes]:
 *         Results written map to the least significant bit (set) that 
 *         was involved in the intersection
 *         ie: Set 1 2 and 3 intersect, output will be vectors from Set 1
 *         Counts of each vector are printed in parentheses at the end of each vector
 *
 *******************************************************************************/
void writeToFile(float *intersections, float *originalValues){

    ofstream out("result.txt"); //Write output of final intersection to file
    unsigned curPascal = F_SUBSET_COUNT;//Tracks many sets exist in each level of depth(Inital: 1 set at depth 0)   
    unsigned intersectIndex = 1; //An overall count of which intersection is being written

    for(unsigned k = 1; k <= maxDepth; k++){
        for(bitString j = 0; j < curPascal; ++j){
            //A bit pattern showing which sets were invloved in the intersectIndex'th intersection
            bitString bitPattern;

            if(j == 0){
                //Get first pattern of k bits
                bitPattern = (1 << k) -1;
            }else{
                //Get next pattern of k bits
                bitPattern = next_perm(bitPattern);
            }
			//Get the least significant bit from the bitPattern
            bitString setIndex = __builtin_ffs(bitPattern) - 1;
            //Write which set this is, and what bit pattern it maps to
            out << "Set: " << intersectIndex << " Bit pattern: " << bitPattern << 
			    " Least bit: " << setIndex << endl;
            
            for(unsigned i = 0; i < VECTORS_PER_SUBSET * (VECTOR_SIZE + 1); ++i){
                if(intersections[(intersectIndex * VECTORS_PER_SUBSET) + WIDTH * 
                    (i % (VECTOR_SIZE + 1)) + i / (VECTOR_SIZE + 1) ] != minFloat){
                    //If this is the last element, print the vector count from intersections
                    if(i % (VECTOR_SIZE + 1) == (VECTOR_SIZE)){
                        out << "(" << intersections[(intersectIndex * VECTORS_PER_SUBSET) + 
                            WIDTH * (VECTOR_SIZE) + i / (VECTOR_SIZE + 1) ] << ")" << endl;
                    }else{
                        //Write the values, mapped to the original input values
                        out << originalValues[(setIndex * VECTORS_PER_SUBSET) +  
                            (VECTORS_PER_SUBSET * F_SUBSET_COUNT) * (i % (VECTOR_SIZE + 1)) 
                            + i / (VECTOR_SIZE + 1) ] << " ";
                    }
                }
            }
            intersectIndex++;
        }
        //Get the number of sets at the next level of depth
        curPascal = curPascal * ((F_SUBSET_COUNT - k)/ (k + 1.0));
    }
}

/******************************************************************************
 * writeToFile_D
 *
 * Writes the discretized results of intersections to a text file
 *
 * [in]:
 *         *originalValue: an array of the original values read in from the input files
 *         size: The number of sets to be written
 *
 * [out]:
 *         result.txt: A text file containing all of the results from the intersections(discretized)
 *
 * [return]:
 *         void
 *
 * [notes]: The count of how many times a vector appeared in an intersection is written in
 *          parentheses at the end of the vector
 *
 *******************************************************************************/
void writeToFile_D(float *intersections, unsigned size){

    ofstream out("result.txt"); //Write output of final intersection to file

    for(unsigned k = 0; k < size; k++){
        out << "Set " << k << endl;
        for(unsigned i = 0; i < VECTORS_PER_SUBSET * (VECTOR_SIZE + 1); ++i){
            if(intersections[(k * VECTORS_PER_SUBSET) + WIDTH * (i % (VECTOR_SIZE + 1)) + 
                   i / (VECTOR_SIZE + 1)] != minFloat){
                if(i % (VECTOR_SIZE + 1)== (VECTOR_SIZE)){
                    out << "(" << intersections[(k * VECTORS_PER_SUBSET) + WIDTH * 
                        (VECTOR_SIZE) + i / (VECTOR_SIZE + 1) ] << ")" << endl;
                }else{
                    out << intersections[(k * VECTORS_PER_SUBSET) +  WIDTH * 
                        (i % (VECTOR_SIZE + 1)) + i / (VECTOR_SIZE + 1) ] << " ";
                }
            }
        }
    }
}

/******************************************************************************
 * printHelp
 *
 * Prints out all available command parameters, and a short description of each
 *
 * [in]:
 *         None
 * [out]:
 *         A list and description of all command parameters
 *
 * [return]:
 *         void
 *
 *******************************************************************************/
void printHelp(){
    printf("\nCommand Parameters:\n");
    printf("\t-b [int > 0]: Specifies how many bins to discretize into, if discretizing\n");
    printf("\t-c: Instructs program to run all intersections on the CPU\n");
    printf("\t-cg: Instructs program to run all intersections on GPU, then again on CPU (used for testing)\n");
    printf("\t-cores [int >0]: Specifies how many cores to run parallel CPU code on\n");
    printf("\t-d: Instructs program to discretize the input data (Default: 3 bins)\n");
    printf("\t-f [file0 file1 ... fileN]: Manually list all input files to use !!Must be last parameter!!\n");
    printf("\t-fd [int > 0] [file0] [file1]: Specifies to read in X files from exactly 2 file locations, file0 and file1\n");
    printf("\t-gpu [int >= 0]: Specify which device to run GPU segments on. Requires a valid device id\n");
    printf("\t-help: Prints out available command line parameters, then exits program\n");
    printf("\t-in: Instructs program to include intersections within a single family in the final measure calculation (excluded by default)\n");
    printf("\t-md [int > 0]: Specifies maximum depth of intersections to perform. (Default = number of input sets)\n");
    printf("\t-mt: Instructs code to perform check to see if sets to be intersected are empty or not\n");
    printf("\t-o [int > 0]: MANDATORY!! Specifies the number of features per feature vector\n");
    printf("\t-t: Instructs program to time the code, and print results of the timing\n");
    printf("\t-v: Instructs program to print verbose information while running\n");
}

/******************************************************************************
 * cpuIntersections
 *
 * Sets up the algorithm to perform all finite intersections on the CPU
 *
 * [in]:
 *         intersections: A pointer to an array holding all fundamental subsets,
 *             and space for the output of the intersections
 *
 *         prefixPascal: A pointer to an array of prefix summed Pascal Numbers used
 *             to determine how many intersections to perform
 *
 *         time_code: Boolean determining whether to time the code or not
 * [out]:
 *         intersections: The results of all intersections saved to the array
 *
 * [return]:
 *         void
 *
 *******************************************************************************/
void cpuIntersections(float* intersections, float* prefixPascal, bool time_code){
    pthread_t* threads = new pthread_t[CORES];
    void *status;

    printf("Performing CPU (%i Cores) Power Set on %i Fundamental Subsets\n",CORES, F_SUBSET_COUNT);
    //Time and perform the intersections on the CPU   
    clock_t st = clock();
    
    float curPascal = F_SUBSET_COUNT;
    for(unsigned i = 1; i < maxDepth; ++i){
        cpuDepth++;
        //Get next Pascal number
        curPascal = curPascal * ((F_SUBSET_COUNT - i) / (i + 1.0));
        bitPermute = (1 << i + 1) - 1;
        bitCount = 0;
        
        for(unsigned j = 0; j < CORES; ++j){
            //Prepare the parameters for the intersect CPU thread function
            intersectArgs* args = (intersectArgs*)malloc(sizeof(args));
            args->a = intersections;
            args->pascal = curPascal;
            args->prefixes = prefixPascal;
            //Call intersect function to perform an intersection on 
            // the sets #leastBit and #(bitPattern-leastBit)
            pthread_create(&threads[j], NULL, intersectCPU, args);
        }
        for(unsigned j = 0; j < CORES; ++j){
            pthread_join(threads[j], &status);
        }
    }

    //End timing and print out runtime
    if(time_code == true){
        clock_t ed = clock();
        clock_t stm = clock();
        clock_t edm = clock();
        cout << "Elapsed time on host: "<<(((float)((ed - st) / CORES) + (edm - stm) ) / 
            (float)CLOCKS_PER_SEC) * 1000    << " ms" << std::endl;
    }
}

/******************************************************************************
 * gpuIntersections
 *
 * Sets up and launches the kernels that perform intersections on the GPU
 *
 * [in]:
 *         intersections: A pointer to an array holding all fundamental subsets,
 *             and space for the output of the intersections
 *
 *         prefixPascal: A pointer to an array of prefix summed Pascal Numbers used
 *             to determine how many intersections to perform
 *
 *         time_code: Boolean determining whether to time the code or not
 *
 *         emptySetSize: Determines how many sets will result from the finite intersections
 *
 * [out]:
 *         intersections: The results of all intersections saved to the array
 *
 * [return]:
 *         void
 *
 *******************************************************************************/
void gpuIntersections(float* intersections, float* prefixPascal, bool time_code, 
                          unsigned emptySetSize){
    //A set to track which sets are empty/non-empty,
    float *emptySets = new float[emptySetSize];
    float *deviceEmptySets;
    for(bitString i = 0; i < emptySetSize; ++i){
        emptySets[i] = minFloat;
    }

    if(emptySetCheck == true){
        CUDA_CHECK_RETURN(hipMalloc((void**)&deviceEmptySets, emptySetSize*sizeof(float)));
        CUDA_CHECK_RETURN(hipMemcpy(deviceEmptySets, emptySets, emptySetSize*sizeof(float),
        hipMemcpyHostToDevice));
    }

    printf("Performing GPU Power Set on %i Fundamental Subsets\n", F_SUBSET_COUNT);

    //Declare array to track which sets in intersections set are not empty sets
    //Set up timer code
    hipEvent_t start, stop;
    float elapsedTime;
    hipStream_t* streams = new hipStream_t[STREAMS];
    if(time_code == true){
        CUDA_CHECK_RETURN(hipEventCreate(&start));
        CUDA_CHECK_RETURN(hipEventCreate(&stop));
        CUDA_CHECK_RETURN(hipEventRecord(start, 0));
    }

    float curPascal = F_SUBSET_COUNT;
    for(unsigned j = 0; j < STREAMS; ++j){
        CUDA_CHECK_RETURN(hipStreamCreate(&streams[j])); //Create streams
    }

    for(unsigned i = 1; i < maxDepth; ++i){
        //Use pascal numbers to determine how many intersections are performed at this level
        curPascal = curPascal * ((F_SUBSET_COUNT - i)/ (i + 1.0));
        bitString bitPattern = (1 << i + 1) -1; //Get the first combination of i bits

        if(emptySetCheck == true){
            //For every intersection of i sets
            CUDA_CHECK_RETURN(hipMemcpy(emptySets, deviceEmptySets, emptySetSize*
                                 sizeof(float), hipMemcpyDeviceToHost));
            CUDA_CHECK_RETURN(hipGetLastError());
        }
        for(bitString j = 0; j < curPascal; ++j){
            bitString leastBit = bitPattern & -bitPattern;
            bitString setAIndex = prefixPascal[i] + 
                getBitPatternIndex(i, bitPattern - leastBit);
            //Check if the sets to be intersected are empty
            if(emptySetCheck == false || emptySets[setAIndex] > minFloat || i == 1){
                unsigned myStream = j % STREAMS;
                //Launch intersection into stream j
                intersectKernel <<< 
                                   1 + (VECTORS_PER_SUBSET / TILE_WIDTH), TILE_WIDTH, 
                                   TILE_WIDTH * (VECTOR_SIZE + 1) * sizeof(float) * 2, 
                                   streams[myStream] 
                                >>>
                                (
                                    intersections, F_SUBSET_COUNT, VECTORS_PER_SUBSET, 
                                    VECTOR_SIZE, 1 + getBitPatternIndex(1, leastBit), setAIndex, 
                                    prefixPascal[i+1] + j, minFloat, WIDTH
                                );
                CUDA_CHECK_RETURN(hipGetLastError());

                if(emptySetCheck == true){
                    //Determine if the intersection performed yielded the empty set
                    isEmptyKernel <<< 
                                     (unsigned)ceil((float)VECTORS_PER_SUBSET / 
                                         min(VECTORS_PER_SUBSET, 512)), 
                                     min(VECTORS_PER_SUBSET, 512), 0, streams[myStream] 
                                  >>>
                                  (
                                     intersections, deviceEmptySets, prefixPascal[i+1] + j, 
                                     VECTORS_PER_SUBSET, minFloat
                                  );
                }
            }
            //Get the next combination of bits
            bitPattern = next_perm(bitPattern);
        }
        hipDeviceSynchronize();
    }

    //Destroy all streams
    for(unsigned j = 0; j < STREAMS; ++j){
        CUDA_CHECK_RETURN(hipStreamDestroy(streams[j]));
    }
    hipDeviceSynchronize();

    if(time_code == true){
        CUDA_CHECK_RETURN(hipDeviceSynchronize());// Wait for the GPU launched work to complete

        CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
        CUDA_CHECK_RETURN(hipEventSynchronize(stop));
        CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));

        CUDA_CHECK_RETURN(hipEventDestroy(start));
        CUDA_CHECK_RETURN(hipEventDestroy(stop));
        cout << "Elapsed kernel time: " << elapsedTime << " ms" << std::endl;
    }
    CUDA_CHECK_RETURN(hipGetLastError());
    hipFree(deviceEmptySets);
}

/******************************************************************************
 * DeTopS main
 *
 * F_SUBSET_COUNT: The specified number of fundamental subsets the data is to be divided into
 * VECTOR_SIZE:    The specified number of elements each feature vector contains
 * VECTORS_PER_SUBSET: The specified number of feature vectors in a fundamental subset
 *
 * totalSize: The total number of elements in the input data
 * fundamentalSubset: A float array that holds the input data
 * intersections: The descriptions of the fundamental subsets, and all descriptive intersections 
 *                between them. In the case a set description's size < the set's size, the extra 
 *                space for that subset is filled with minimum float values
 *
 * [Command Line parameters]:
 *         discretize_input:
 *             Determines whether the input data will be discretized
 *             Requires -b if used
 *             Default: False
 *             Set true by command param -d
 *
 *         num_bins:
 *             Determines how many discrete values are to be used when discretizing
 *             Default: 3
 *             Set by command param -b [int>0]
 *
 *         inputFile:
 *             Determines which files the data is to be read from
 *             !Must be the last parameter entered, followed only by the input files!
 *             Use either this or -fd
 *             Default: None
 *             Set by command param -f [file1 file2 ... fileN]
 *
 *        useCPU:
 *            Determine whether the intersections will be performed on CPU or GPU
 *            Default: False (Run intersections on GPU)
 *            Set true by command param
 *                -c (Sets useGPU false) or
 *                -cg (Sets useCPU and useGPU true)
 *
 *        emptySetCheck:
 *            Determine whether the GPU will check if a set is empty before performing intersection
 *            Setting true may speed up or slow down results, 
 *                 depending on the data, but the output will be the same
 *            Default: False
 *            Set True by command param -mt
 *
 *        time_code:
 *            Determines whether the program will be timed while running or not
 *            Default: False
 *            Set true by command param -t
 *
 *        VECTOR_SIZE:
 *            Determines the number of elements in each feature vector
 *            !Mandatory!
 *            Must be a whole number > 0
 *            Set by command param -o [int>0]
 *
 *        CORES:
 *            Determines how many cores the cpu will run on
 *            Default: 1
 *            Set by command param -cores [int>0]
 *
 *        verbose_info:
 *            Determines if detailed output will be printed
 *            This includes:
 *                Measure for each intersection
 *                Device information
 *                Number of unique feature vectors in each fundamental subset
 *            Default: False
 *            Set true by command param -v
 *        
 *        measure_within_set:
 *            Specify weather measure should be calculated for intersections of sets from one family
 *            Default: False (do not include these in the measure)
 *            Set true by command param -in
 *              
 *        device:
 *            Determines which device the GPU code will run on
 *            Takes in the integer id of a CUDA device
 *            Default: 0
 *            Set by command param -gpu [int>0]
 *
 *        set1, set2:
 *            Determines where to read files from
 *            Takes in an integer followed by two strings
 *            Integer is how many sets to read
 *            String 1 is the file path and name of first set, minus the number
 *            String 2 is the file path and name of second set, minus the number
 *            Use either this or -f
 *            Set by command param -fd [int>0] [string] [string]
 *
 *
 * Input Assumptions:
 *    Each input file will represent 1 Fundamental Subset
 *    Each input file will have the same dimensions 
 *         (Vectors per subset, features per vector, total size)
 *    User will provide the number of elements per feature vector at run time
 *
 *******************************************************************************/
int main(int argc, const char ** argv) {

    bool gpuDevice = false; //Tracks whether a GPU device is available or not
    bool useCPU = false;    //Perform calculations on CPU?
    bool useGPU = true;     //Perform calculations on GPU? (default)

//--------------------------------------------------------------------------------------------------
//This section of code deals with input parameters from the command line

    //Initialize default option values
    unsigned device = 0; //ID of GPU to run on 
    bool discretize_input = true; //Discretize the input files
    bool discrete_output = false; //Discretize the output files
    unsigned num_bins = 15; //Discrete false by default, this is for simplifying testing
    bool time_code = false; //Time the code
    bool verbose_info = false; //Print calculation details
    bool measure_within_set = false; //Include or exclude single family intersections
    
    std::string file_pattern; //Name of files for the input data
    int setA_index; //Index of sets for set family A
	int setB_index; //Index of sets for set family B
    std::vector<std::string> fileName; //Store list of input files

    //Set option values for each parameter entered
    for(unsigned i = 0; i < argc; ++i){
        if(argv[i] == std::string("-c")){
            //Set program to perform on CPU only
            useCPU = true;
            useGPU = false;

        }else if(argv[i] == std::string("-cg")){
            //Set program to perform on GPU then CPU
            useCPU = true;

        }else if(argv[i] == std::string("-d")){
            //Instruct program to discretize input
            discretize_input = true;

        }else if(argv[i] == std::string("-b")){
            //Set how many bins to discretize into
            std::stringstream convert(argv[i + 1]);
            convert >> num_bins;
            i++;

        }else if(argv[i] == std::string("-mt")){
            //Indicate whether the GPU code should check for empty sets or not
            emptySetCheck = true;

        }else if(argv[i] == std::string("-gpu")){
            //Which device to use
            std::stringstream convert(argv[i + 1]);
            convert >> device;
            i++;

        }else if(argv[i] == std::string("-t")){
            time_code = true;

        }else if(argv[i] == std::string("-cores")){
            //How many cores the cpu has
            std::stringstream convert(argv[i + 1]);
                convert >> CORES;
                if(CORES < 1)
                    CORES = 1;
            i++;

        }else if(argv[i] == std::string("-o")){
            //Declare the size of the feature vectors
            std::stringstream convert(argv[i + 1]);
            convert >> VECTOR_SIZE;
            i++;

        }else if(argv[i] == std::string("-md")){
            //Declare the maximum depth of intersections
            std::stringstream convert(argv[i + 1]);
            convert >> maxDepth;
            i++;

        }else if(argv[i] == std::string("-v")){
            verbose_info = true;

        }else if(argv[i] == std::string("-do")){
            discrete_output = true;

        }else if(argv[i] == std::string("-fd")){
            std::stringstream convert(argv[i + 1]);
			convert >> SETS;
			std::stringstream convertA(argv[i + 3]);
			convertA >> setA_index;
			std::stringstream convertB(argv[i + 4]);
			convertB >> setB_index;
            file_pattern = argv[i+2];
            i+=4;

        }else if(argv[i] == std::string("-help")){
            printHelp();
            return(0);
        
        }else if(argv[i] == std::string("-in")){
            measure_within_set = true;
            
        }else if(argv[i] == std::string("-f")){
            //Push all files after -f into fileName vector
            for(unsigned j = i + 1; j<argc; ++j){
                fileName.push_back(argv[j]);
            }
            break;

        }else if(i > 0){
            std::cout << "Unknown parameter " << argv[i] << 
                ", use -help for a list of possible parameters.\n";
        }
    }

    //Check for valid VECTOR_SIZE
    if(VECTOR_SIZE < 1){
        std::cerr << "The number of elements in each feature vector must be > 0. " <<
            "Set this with the -o parameter.\n";
        exit(1);
    }

    if(fileName.size() == 0){
        string fileNumber;
		
		for(unsigned i = 0; i < SETS/2; i++){
			std::string fileString = file_pattern;
            ostringstream convert;
            convert << (setA_index + i);
            fileNumber = convert.str();
            fileName.push_back(fileString.append(fileNumber).append(".txt"));
        }
		
		for(unsigned i = 0; i < SETS/2; i++){
			std::string fileString = file_pattern;
            ostringstream convert;
            convert << (setB_index + i);
            fileNumber = convert.str();
            fileName.push_back(fileString.append(fileNumber).append(".txt"));
        }
    }

    //Number of Fundamental Subsets is equal to the number of input files
    F_SUBSET_COUNT = fileName.size();

    //Throw error if discretize option is chosen, but invalid, or no bin count is supplied
    if(discretize_input == true && num_bins < 1){
            std::cerr << "Use of -d requires -b  > 0\n";
            exit(1);
    }

    //If maxDepth wasn't set (or is invalid) set to F_SUBSET_COUNT (max depth)
    if(maxDepth <= 0 || maxDepth > F_SUBSET_COUNT)
        maxDepth = F_SUBSET_COUNT;

//--------------------------------------------------------------------------------------------------
    CUDA_CHECK_RETURN(hipSetDevice(device));

    size_t deviceMemory;
    //Get information about the available devices
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for(int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        if(verbose_info == true){
            printf("Device Number: %d\n", i);
            printf("Device name: %s\n", prop.name);
            printf("Shared Memory Limit: %d\n", prop.sharedMemPerBlock);
            printf("Potential tile width: %f!\n", min( 512.0,pow((float)2,
                      floor(log2f(prop.sharedMemPerBlock / ((VECTOR_SIZE + 1)*sizeof(float)*2))))));
        }
        //prop.major checks if the device is emulated or not. 
        //If not emulated, a gpu device is available
        if(prop.major != 9999) 
            gpuDevice = true;
        //Set TILE_WIDTH to the max the specified device can handle
        if(device == i){
            TILE_WIDTH = min(512.0,pow((float)2,floor(log2f(prop.sharedMemPerBlock / 
                             ((VECTOR_SIZE + 1) * sizeof(float) * 2)))));
            deviceMemory = prop.totalGlobalMem;
        }
    }
    std::cout << "Using device " << device << std::endl;
    
    //If no GPU is available, use the CPU
    if(gpuDevice==false) 
        useCPU = true;

    //Total number of elements across all Fundamental Subsets, does not include the Count attribute
    unsigned totalSize = 0;

    std::fstream myfile(fileName[0].c_str(), std::ios_base::in);

    if(myfile.fail()){
        cerr << "Error: " << fileName[0].c_str() << " could not be found.\n";
        exit(1);
    }

    float fileElement;
    //Get size of file
    while (myfile >> fileElement) {
        totalSize++;
    }
    //Get total size of all files combined
    totalSize *= F_SUBSET_COUNT;
    myfile.close();

    //Number of Feature Vectors in each Fundamental Subset
    VECTORS_PER_SUBSET = (totalSize / VECTOR_SIZE) / F_SUBSET_COUNT;

    if(verbose_info == true)
        printf("(Vectors per Set %i)  (Total Size %i) (Vector Size %i)  (Set Count %i)\n", 
                   VECTORS_PER_SUBSET, totalSize/F_SUBSET_COUNT, VECTOR_SIZE, F_SUBSET_COUNT);

    //Calculate how many sets can be handles, given the size of each, and desired intersection depth
    deviceMemory = deviceMemory / (VECTORS_PER_SUBSET * (VECTOR_SIZE+1) * sizeof(float));
    unsigned possibleSets = 1;
    while(true){
        unsigned x = 1;
        bitString pascalSum = 1;
        for(unsigned i = 0; i < maxDepth; ++i){
            if(i>possibleSets) break;
            x = x * ((possibleSets + 1 - i) / (i + 1.0));
            pascalSum += x;
        }
        if(pascalSum > deviceMemory)
            break;
        possibleSets++;
    }

    //Lower possible sets to account for other data usage
    possibleSets = possibleSets - (2 - (possibleSets % 2));
    //Max possible sets is 64 (# of bits in an unsigned long long int)
    possibleSets = min(64, possibleSets);

    printf("With depth %i, you can handle %i sets!\n",maxDepth, possibleSets);

    //If user is trying to run too many files, exit program
    if(F_SUBSET_COUNT > possibleSets){
        std::cerr << "Not enough memory for " << F_SUBSET_COUNT <<" sets of " << VECTOR_SIZE << 
            "x" << VECTORS_PER_SUBSET << " elements\n";
        exit(1);
    }

    //Get pascalMax, the highest pascal number of the F_SUBSET_COUNT-th row,
    // pascalMax also stores how many parallel streams are needed
    //Get pascalTotal, a weighted sum of pascal numbers, used to calculate final measure
    //Initialize totalMeasure, which holds the sum of all weighted measures
    float *prefixPascal = new float[maxDepth+1];
    prefixPascal[0] = 0;
    prefixPascal[1] = 1;
    bitString pascalMax = 0;
    float totalMeasure = 0;
    bitString pascalTotal = F_SUBSET_COUNT;
    bitString emptySetSize = 1;

    float x = F_SUBSET_COUNT;
    for(unsigned i = 1; i <= maxDepth; ++i){
        emptySetSize += x;
        if(i < maxDepth){
            prefixPascal[i+1] = x + prefixPascal[i];
        }
        if(x > pascalMax)
            pascalMax = x;
        
        x = x * ((F_SUBSET_COUNT - i)/ (i + 1.0));
        if(i<maxDepth){
            pascalTotal += (1 + i) * (1 + i) * x;
        }
    }

    //Total width of the intersections power set array
    WIDTH = emptySetSize * VECTORS_PER_SUBSET;

    //Declare array to be discretized, of size, filesize
    float *fundamentalSubset = new float[totalSize];
    float *originalValues;
    if(discrete_output == false)
        originalValues = new float[totalSize];

    //Fill in array with values from input file
    for(unsigned i = 0; i < F_SUBSET_COUNT; ++i){
        unsigned z = 0;
        std::fstream inputFile(fileName[i].c_str(), std::ios_base::in);
        if(inputFile.fail()){
            cerr << "Error: File " << fileName[i].c_str() << " could not be found.\n";
            exit(1);
        }

        while (inputFile >> fileElement) {
            fundamentalSubset[(i * VECTORS_PER_SUBSET) + (z / VECTOR_SIZE) + ((z % VECTOR_SIZE) * 
                VECTORS_PER_SUBSET * F_SUBSET_COUNT)] = fileElement;
            //If user wants original values output, store them in a seperate array
            if(discrete_output == false)
                originalValues[(i * VECTORS_PER_SUBSET) + (z / VECTOR_SIZE) + ((z % VECTOR_SIZE) * 
                    VECTORS_PER_SUBSET * F_SUBSET_COUNT)] = fileElement;
            z++;
        }
        z = 0;
    }

    //Call function to discretize, if specified by user
    if(discretize_input == true)
        discretize(fundamentalSubset, totalSize, num_bins);
        
    //Declare array to hold the Set Descriptions of each Fundamental Subset
    bitString interSetSize = WIDTH * (VECTOR_SIZE + 1);
    float *intersections = new float[interSetSize];
    CUDA_CHECK_RETURN(hipMallocManaged(&intersections, interSetSize * sizeof(float)));
    printf("Malloc Intersection Set: %s \n", hipGetErrorString(hipGetLastError()));

    //Jump here after GPU calculation, if performing calculation on both CPU and GPU (for testing)
    rerunOnCPU:

    //Set all values in the Set Description array to the minimum float value
    initNegative(intersections, WIDTH * (VECTOR_SIZE + 1));

    //Create the Set Descriptions and save them into the Set Description array
    createSetDescription(fundamentalSubset, intersections);

    //Run the intersections on the GPUs
    //If no GPUs are found, or the user specifies to not use them, run the intersections on the CPU
    if(useGPU == true){
        gpuIntersections(intersections, prefixPascal, time_code, emptySetSize);
    }

    if(useCPU == true){
        if(useGPU == true){
            //If user instructed to use CPU and GPU, clear results and run CPU code
            useGPU = false;
            goto rerunOnCPU;
        }
        cpuIntersections(intersections, prefixPascal, time_code);
    }

    //Calculates the final measure of the closeness of intersections
    totalMeasure = calculateMeasure(emptySetSize, prefixPascal, intersections, pascalTotal, 
                       verbose_info, measure_within_set);

    //Write output of final intersection to file
    if(discrete_output == true){
        writeToFile_D(intersections, emptySetSize);
    }else{
        writeToFile(intersections, originalValues);
    }
    
    //Print final measure of nearness of sets
    printf("\nDescriptive Set Intersections final Measure: %f\n", totalMeasure);
	
    hipFree(intersections);
    return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line, 
                                  const char *statement, hipError_t err){
    if (err == hipSuccess)
        return;
    std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at " 
        << file << ":" << line << std::endl;
    exit (1);
}
